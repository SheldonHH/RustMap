/*
This code is a convenience tool for profiling the CUDA kernels in the training
loop of train_gpt2.cu. Compile:

make profile_gpt2cu NO_MULTI_GPU=1

And then e.g. use ncu from NVIDIA. The CLI docs for example:
https://docs.nvidia.com/nsight-compute/NsightComputeCli/

TLDR run like:

sudo ncu --set full --import-source yes -o profile -f ./profile_gpt2cu

This:
- `--set full` means we'll collect A LOT of metrics. take out for less
- `--import-source yes` means we'll get the source code in the profile
- `-o profile` writes the results into file profile.ncu-rep
- `-f` forces overwrite of the profile.ncu-rep file
- `./profile_gpt2cu` is the executable we want to profile

This writes results into profile.ncu-rep output file.
You can open this up in NVIDIA Nsight Compute UI.
For example, I have NVIDIA Nsight Compute installed on my Mac, and I rsync
the profile.ncu-rep from a cloud box to local to pretty view.
*/

#define TESTING
#include "train_gpt2.cu"

int main() {

    // set up the device
    int deviceIdx = 0;
    cudaCheck(hipSetDevice(deviceIdx));
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deviceIdx);
    cuda_num_SMs = deviceProp.multiProcessorCount;
    printf("[System]\n");
    printf("Device %d: %s\n", deviceIdx, deviceProp.name);

    // setup cuBLAS and cuBLASLt
    cublasCheck(hipblasCreate(&cublas_handle));
    cublasCheck(hipblasLtCreate(&cublaslt_handle));
    // TF32 precision is equivalent to torch.set_float32_matmul_precision('high')
    int enable_tf32 = deviceProp.major >= 8 ? 1 : 0;
    printf("enable_tf32: %d\n", enable_tf32);
    cublas_compute_type = enable_tf32 ? HIPBLAS_COMPUTE_32F_FAST_TF32 : HIPBLAS_COMPUTE_32F;
    hipblasMath_t cublas_math_mode = enable_tf32 ? HIPBLAS_TF32_TENSOR_OP_MATH : HIPBLAS_DEFAULT_MATH;
    cublasCheck(hipblasSetMathMode(cublas_handle, cublas_math_mode));
    // setup the (global) cuBLASLt workspace
    cudaCheck(hipMalloc(&cublaslt_workspace, cublaslt_workspace_size));

    #ifdef ENABLE_CUDNN
    checkCudnnErr(hipdnnCreate(&cudnn_handle));
    #endif

    // build the GPT-2 model from a checkpoint
    GPT2 model;
    gpt2_build_from_checkpoint(&model, "gpt2_124M_bf16.bin");

    int B = 4;
    int T = 1024;
    printf("batch size: %d\n", B);
    printf("sequence length: %d\n", T);

    int* x = (int*)mallocCheck(B * T * sizeof(int));
    int* y = (int*)mallocCheck(B * T * sizeof(int));
    for(int  i = 0; i < B  * T; ++i) {
        x[i] = i % model.config.vocab_size;
        y[i] = i % model.config.vocab_size;
    }

    model.config.num_layers = 1;

    // do a training step
    gpt2_forward(&model, x, y, B, T);
    gpt2_zero_grad(&model);
    gpt2_backward(&model);
    gpt2_update(&model, 1e-4f, 0.9f, 0.999f, 1e-8f, 0.0f, 1);
    cudaCheck(hipDeviceSynchronize()); // finish all CUDA work to get correct precise timings
    // free
    gpt2_free(&model);

    #ifdef ENABLE_CUDNN
    if (cudnn_workspace != NULL) { cudaCheck(hipFree(cudnn_workspace)); }
    checkCudnnErr(hipdnnDestroy(cudnn_handle));
    #endif
    cudaCheck(hipFree(cublaslt_workspace));
    cublasCheck(hipblasDestroy(cublas_handle));
    cublasCheck(hipblasLtDestroy(cublaslt_handle));

    return 0;
}
