#include "hip/hip_runtime.h"
#define TESTING
#include "train_gpt2.cu"

// poor man's tensor checker
int check_tensor(float *a, float *b, int n, const char* label, float threshold=1e-0) {
    // a is the calculated tensor, b is the reference tensor
    int print_upto = 10;
    int ok = 1;
    float max_diff = 0.0f;
    float max_rel_error = 0.0f;
    float max_a = 0.0f;
    float max_b = 0.0f;
    printf("%s\n", label);
    for (int i = 0; i < n; i++) {
        float diff = fabsf(a[i] - b[i]);
        if (diff > max_diff) {
            max_diff = diff;
            float denom = fabsf(b[i]);
            max_rel_error = (denom == 0.0f) ? 0.0f : diff / denom;
            max_a = a[i];
            max_b = b[i];
        }
        if (diff <= threshold) {
            if (i < print_upto) { printf("OK "); }
        } else {
            if (i < print_upto) { printf("NOT OK "); }
            ok = 0;
        }
        if (i < print_upto) { printf("%f %f\n", a[i], b[i]); }
    }
    // print the final result
    if (ok) {
        printf("TENSOR OK, max diff: %e, with rel error: %e (calculated=%f, ref=%f)\n",
                max_diff, max_rel_error, max_a, max_b);
    } else {
        printf("TENSOR NOT OK, max diff: %e, with rel error: %e (calculated=%f, ref=%f)\n",
                max_diff, max_rel_error, max_a, max_b);
    }
    return ok;
}

// the same tensors as in the train file, but in float, which are used as reference
typedef struct {
    float*  wte; // (Vp, C)
    float*  wpe; // (maxT, C)
    float*  ln1w; // (L, C)
    float*  ln1b; // (L, C)
    float*  qkvw; // (L, 3*C, C)
    float*  qkvb; // (L, 3*C)
    float*  attprojw; // (L, C, C)
    float*  attprojb; // (L, C)
    float*  ln2w; // (L, C)
    float*  ln2b; // (L, C)
    float*  fcw; // (L, 4*C, C)
    float*  fcb; // (L, 4*C)
    float*  fcprojw; // (L, C, 4*C)
    float*  fcprojb; // (L, C)
    float*  lnfw; // (C)
    float*  lnfb; // (C)
} FloatParameterTensors;
static_assert(sizeof(FloatParameterTensors) == NUM_PARAMETER_TENSORS * sizeof(void*), "Inconsistent sizes!");

// malloc_and_point, but in float and on CPU, because we use this data to check correctness on CPU
float* float_cpu_malloc_and_point_parameters(FloatParameterTensors* params, size_t* param_sizes) {
    // calculate the total number of parameters
    size_t num_parameters = 0;
    for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        num_parameters += param_sizes[i];
    }
    // everything is float so number of bytes to allocate is a simple multiplication
    float* params_memory = (float*)mallocCheck(num_parameters * sizeof(float));
    float** ptrs[] = {
        &params->wte, &params->wpe, &params->ln1w, &params->ln1b, &params->qkvw, &params->qkvb,
        &params->attprojw, &params->attprojb, &params->ln2w, &params->ln2b, &params->fcw, &params->fcb,
        &params->fcprojw, &params->fcprojb, &params->lnfw, &params->lnfb
    };
    float* params_memory_iterator = params_memory;
    for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        *(ptrs[i]) = params_memory_iterator;
        params_memory_iterator += param_sizes[i];
    }
    return params_memory;
}

int main(int argc, char *argv[]) {

    // set up the device
    int deviceIdx = 0;
    cudaCheck(hipSetDevice(deviceIdx));
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deviceIdx);
    cuda_num_SMs = deviceProp.multiProcessorCount;
    cuda_arch_major = deviceProp.major;
    cuda_arch_minor = deviceProp.minor;
    printf("[System]\n");
    printf("Device %d: %s\n", deviceIdx, deviceProp.name);

    // setup cuBLAS and cuBLASLt
    cublasCheck(hipblasCreate(&cublas_handle));
    cublasCheck(hipblasLtCreate(&cublaslt_handle));
    // TF32 precision is equivalent to torch.set_float32_matmul_precision('high')
    int enable_tf32 = cuda_arch_major >= 8 ? 1 : 0;
    enable_tf32 = 0; // NOTE: disable TF32 for testing!!!
    printf("enable_tf32: %d\n", enable_tf32);
    cublas_compute_type = enable_tf32 ? HIPBLAS_COMPUTE_32F_FAST_TF32 : HIPBLAS_COMPUTE_32F;
    hipblasMath_t cublas_math_mode = enable_tf32 ? HIPBLAS_TF32_TENSOR_OP_MATH : HIPBLAS_DEFAULT_MATH;
    cublasCheck(hipblasSetMathMode(cublas_handle, cublas_math_mode));
    cudaCheck(hipMalloc(&cublaslt_workspace, cublaslt_workspace_size));

    #ifdef ENABLE_CUDNN
    checkCudnnErr(hipdnnCreate(&cudnn_handle));
    #endif

    // build the GPT-2 model from a checkpoint
    GPT2 model;
    gpt2_build_from_checkpoint(&model, load_filename);
    size_t V = model.config.vocab_size;
    size_t Vp = model.config.padded_vocab_size;
    size_t maxT = model.config.max_seq_len;
    size_t L = model.config.num_layers;
    size_t C = model.config.channels;

    // load additional information that we will use for debugging and error checking
    FILE *state_file = fopenCheck("gpt2_124M_debug_state.bin", "rb");
    int state_header[256];
    freadCheck(state_header, sizeof(int), 256, state_file);
    if (state_header[0] != 20240327) { fprintf(stderr, "Bad magic state file\n"); exit(EXIT_FAILURE); }
    if (state_header[1] != 2) {
        fprintf(stderr, "Bad version in state file\n");
        fprintf(stderr, "---> HINT: try to re-run `python train_gpt2.py`\n");
        exit(EXIT_FAILURE);
    }
    int B = state_header[2]; // batch size, e.g. 4
    int T = state_header[3]; // time / sequence length (e.g. 64, up to maxT)
    assert(0 <= T && T <= maxT);
    printf("[State]\n");
    printf("batch_size: %d\n", B);
    printf("seq_len: %d\n", T);

    // read reference information from the file saved from Python/PyTorch side
    // 1) input x and y
    int* x = (int*)mallocCheck(B * T * sizeof(int));
    int* y = (int*)mallocCheck(B * T * sizeof(int));
    freadCheck(x, sizeof(int), B*T, state_file);
    freadCheck(y, sizeof(int), B*T, state_file);
    // 2) results of forward pass (logits and loss)
    float* expected_logits = (float*) mallocCheck(B * T * V * sizeof(float));
    float* expected_loss = (float*) mallocCheck(1 * sizeof(float));
    freadCheck(expected_logits, sizeof(float), B*T*V, state_file);
    freadCheck(expected_loss, sizeof(float), 1, state_file);
    // 3) results of backward pass (parameter gradients)
    FloatParameterTensors expected_grads; // will be read from file. right now: all in fp32
    float* expected_grads_memory = float_cpu_malloc_and_point_parameters(&expected_grads, model.param_elements);
    freadCheck(expected_grads_memory, sizeof(float), model.num_parameters, state_file);
    fcloseCheck(state_file);

    // this memory will be used to do one single copy of all (mixed precision) GPU grads to CPU grads
    void* grads_memory_cpu = mallocCheck(model.num_parameters_bytes);
    float* grads_memory_cpu_float = (float*)mallocCheck(model.num_parameters * sizeof(float));

    // overall OK signal for the test
    int allok = 1;

    // First, do target-free forward pass to validate logits
    gpt2_forward(&model, x, NULL, B, T);
    // at this point, target should be equal to expected_logits, let's compare
    // copy logits to CPU so we can compare them
    floatX* logits_cpu_raw = (floatX*)mallocCheck(B * T * Vp * sizeof(floatX));
    float* logits_cpu = (float*)mallocCheck(B * T * Vp * sizeof(float));
    hipMemcpy(logits_cpu_raw, model.acts.output, B * T * Vp * sizeof(floatX), hipMemcpyDeviceToHost);
    for (int i = 0; i < B * T * Vp; i++) {
        logits_cpu[i] = (float)logits_cpu_raw[i];
    }

    // FP16 and lower require very high tolerances unfortunately. TODO look into more
    float logit_accuracy_threshold = 1e-2f;
    float loss_diff_threshold = 0.05f;
    #if defined(ENABLE_BF16) || defined(ENABLE_F16)
    logit_accuracy_threshold = 25.0f; // 15.0f was too low even without cuDNN?! :(
    #endif

    // compare the output logits from the forward pass
    // also careful that we don't access and compare the padded columns of logits
    int logits_ok = 1;
    float max_diff = 0.0f;
    for (int bt = 0; bt < B*T; bt++) {
        for (int v = 0; v < V; v++) {
            int i = bt * Vp + v; // linearized index
            if (i < 10) {
                printf("%f, %f\n", expected_logits[i], logits_cpu[i]);
            }
            float diff = fabsf(expected_logits[bt*V + v] - logits_cpu[i]);
            max_diff = fmaxf(max_diff, diff);
            if (diff >= logit_accuracy_threshold) {
                printf("MISMATCH AT INDEX %d,%d: ", bt, v);
                printf("%f %f\n", expected_logits[bt*V + v], logits_cpu[i]);
                logits_ok = 0;
                bt = B*T; // to break out of both loops
                break;
            }
        }
    }
    allok = allok && logits_ok;
    if(!logits_ok) { printf("NOT "); }
    printf("OK (LOGITS)\n");
    printf("logit max diff: %f\n", max_diff);

    // let's do 10 training iterations, following the pytorch code
    float losses[10];
    for (int step = 0; step < 10; step++) {
        struct timespec start, end;
        clock_gettime(CLOCK_MONOTONIC, &start);
        gpt2_forward(&model, x, y, B, T);
        gpt2_zero_grad(&model);
        gpt2_backward(&model);
        clock_gettime(CLOCK_MONOTONIC, &end);
        double time_elapsed_s = (end.tv_sec - start.tv_sec) + (end.tv_nsec - start.tv_nsec) / 1e9;

        if (step == 0) {
            // error checking at step 0 for reference activations

            // compare the achieved loss
            if (fabsf(model.mean_loss - *expected_loss) >= loss_diff_threshold) {
                printf("LOSS MISMATCH: %f %f\n", model.mean_loss, *expected_loss);
                allok = 0;
            } else {
                printf("LOSS OK: %f %f\n", model.mean_loss, *expected_loss);
            }

            // move the (mixed precision) grads from GPU to CPU
            hipMemcpy(grads_memory_cpu, model.grads_memory, model.num_parameters_bytes, hipMemcpyDeviceToHost);

            // convert all gradients to float on the CPU
            char* src_iterator = (char*)grads_memory_cpu; // can be lower precision, so we use char*
            float* dst_iterator = (float*)grads_memory_cpu_float; // float*
            float* exp_iterator = expected_grads_memory; // float* of expected gradients from Python
            float* tensors1[NUM_PARAMETER_TENSORS];
            float* tensors2[NUM_PARAMETER_TENSORS];
            for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
                if (model.param_sizeof[i] == sizeof(float)) {
                    // float tensor => copy over directly
                    memcpy(dst_iterator, src_iterator, model.param_elements[i] * sizeof(float));
                } else {
                    // low-precision tensor => convert to float
                    assert(model.param_sizeof[i] == sizeof(floatX)); // floatX is the single non-float supported atm
                    for (size_t j = 0; j < model.param_elements[i]; j++) {
                        dst_iterator[j] = ((floatX*)src_iterator)[j]; // convert to float
                    }
                }
                // for convenience record the position of comparison for reality vs. expectation
                tensors1[i] = dst_iterator; // reality
                tensors2[i] = exp_iterator; // expectation
                // advance the iterators
                src_iterator += model.param_elements[i] * model.param_sizeof[i];
                dst_iterator += model.param_elements[i];
                exp_iterator += model.param_elements[i];
            }

            // compare the gradients on the parameters all at once, in fp32
            // I set the tolerances manually by inspecting the gradient differences for
            // a few elements of each tensor. bf16 looks ok but not amazing here.
            // It's possible we have bugs lurking, or maybe it is bf16. Not 100% sure.
            // Also, if code changes and some of these get tripped, it could be ok if it's not by too much,
            // because our use of stochastic rounding is adding some non-determinism "pepper noise".
            // In that case it's ok to extend the tolerance by a bit, after a manual review.
            allok = allok & check_tensor(tensors1[0], tensors2[0], V * C, "wte", 8e-1f);
            allok = allok & check_tensor(tensors1[1], tensors2[1], maxT * C, "wpe", 1e-2f);
            allok = allok & check_tensor(tensors1[2], tensors2[2], L * 3*C * C, "qkvw", 1.1e-1); // hmm a bit high
            allok = allok & check_tensor(tensors1[3], tensors2[3], L * 3*C, "qkvb", 4e-2f);
            allok = allok & check_tensor(tensors1[4], tensors2[4], L * C * C, "attprojw", 3e-2f);
            allok = allok & check_tensor(tensors1[5], tensors2[5], L * C, "attprojb", 3e-2f);
            allok = allok & check_tensor(tensors1[6], tensors2[6], L * 4*C * C, "fcw", 9e-2f); // hmm a bit high
            allok = allok & check_tensor(tensors1[7], tensors2[7], L * 4*C, "fcb", 9e-2f); // hmm a bit high
            allok = allok & check_tensor(tensors1[8], tensors2[8], L * C * 4*C, "fcprojw", 9e-2f); // hmm a bit high
            allok = allok & check_tensor(tensors1[9], tensors2[9], L * C, "fcprojb", 3e-2f);
            allok = allok & check_tensor(tensors1[10], tensors2[10], L * C, "ln1w", 0.1f); // hmm bit higher
            allok = allok & check_tensor(tensors1[11], tensors2[11], L * C, "ln1b", 3e-2f);
            allok = allok & check_tensor(tensors1[12], tensors2[12], L * C, "ln2w", 0.1f); // hmm bit higher
            allok = allok & check_tensor(tensors1[13], tensors2[13], L * C, "ln2b", 3e-2f);
            allok = allok & check_tensor(tensors1[14], tensors2[14], C, "lnfw", 0.12f); // hmm bit higher
            allok = allok & check_tensor(tensors1[15], tensors2[15], C, "lnfb", 3e-2f);
        }

        gpt2_update(&model, 1e-4f, 0.9f, 0.999f, 1e-8f, 0.01f, step+1);

        // print the timing information at the end
        printf("step %d: loss %f (took %f ms)\n", step+1, model.mean_loss, time_elapsed_s * 1000);
        losses[step] = model.mean_loss;
    }

    // expected losses are as follows, from Python
    float expected_losses[10] = {
        5.270007133483887,
        4.059706687927246,
        3.3751230239868164,
        2.8007826805114746,
        2.315382242202759,
        1.8490285873413086,
        1.3946564197540283,
        0.9991465210914612,
        0.6240804195404053,
        0.37651097774505615
    };

    // compare
    for (int i = 0; i < 10; i++) {
        if (fabsf(losses[i] - expected_losses[i]) >= loss_diff_threshold) {
            printf("LOSS MISMATCH AT STEP %d: %f %f\n", i+1, losses[i], expected_losses[i]);
            allok = 0;
        } else {
            printf("loss ok at step %d: %f %f\n", i+1, losses[i], expected_losses[i]);
        }
    }

    // final approval
    printf("overall okay: %d\n", allok);

    // free everything
    free(x);
    free(y);
    free(logits_cpu_raw);
    free(logits_cpu);
    free(expected_logits);
    free(expected_loss);
    free(expected_grads_memory);
    free(grads_memory_cpu);
    free(grads_memory_cpu_float);
    gpt2_free(&model);
    #ifdef ENABLE_CUDNN
    if (cudnn_workspace != NULL) { cudaCheck(hipFree(cudnn_workspace)); }
    checkCudnnErr(hipdnnDestroy(cudnn_handle));
    #endif
    cudaCheck(hipFree(cublaslt_workspace));
    cublasCheck(hipblasDestroy(cublas_handle));
    cublasCheck(hipblasLtDestroy(cublaslt_handle));

    return 0;
}
