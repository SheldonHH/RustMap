#include "hip/hip_runtime.h"
/*
GPT-2 Transformer Neural Net trained in raw CUDA
Non-trivial notes to be aware of:

We are being clever in the backward pass to conserve memory.
In particular, all parameters use a += in the backward pass, so we
can later do gradient accumulation. But all activations have = instead of +=
because these are faster (just read, no write). This is okay for all activations
except for those in the residual stream, where the gradients have to add. We make
sure that those parts work out ok and that we do a += as necessary. E.g.,
the layernorms are connected to the residuals so we += in layernorm backward.

In this file we are using Mixed Precision training, so different activations,
paramaters, grads and buffers may be kept at different precisions, to take
advantage of the fast low-precision hardware in the latest GPUs (bf16/fp16),
and fp8 (coming soon^TM).

Compile:
make train_gpt2cu

Example launch using bfloat16 on 1 GPU batch size 8, sample/eval every 200 steps:
Also we're using TinyStories here for example as it is a bigger dataset
./train_gpt2cu -b 8 -v 200 -s 200 -i data/TinyStories

Example launch using bfloat16 on 4 GPUs, same as above:
mpirun -np 4 ./train_gpt2cu -b 8 -v 200 -s 200 -i data/TinyStories

If you'd like to see train_gpt2.cu produce identical results to
`python train_gpt2.py`, you can run it like this:
make train_gpt2cu PRECISION=FP32
./train_gpt2cu -b 4 -t 64 -l 1e-4 -v 200 -s 200 -a 1 -x 10 -f 0
This reads & runs in fp32, B=4, T=64, LR=1e-4, val/sample never (200),
-a 1 is "overfit single batch", -x 10 is 10 iterations, and -f 0 disables tf32
*/

#include <stdio.h>
#include <stdlib.h>
#include <stdarg.h>
#include <math.h>
#include <time.h>
#include <assert.h>
#include <float.h>
#include <string.h>
#include <unistd.h>
#include <assert.h>
// GPU / CUDA related
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipblaslt.h>
#include <hip/hip_bf16.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
// Multi-GPU related
#ifdef MULTI_GPU
#include <mpi.h>
#include <nccl.h>
#endif
// our own utilities
// defines: fopenCheck, freadCheck, fcloseCheck, fseekCheck, mallocCheck
#include "utils.h"
// defines: tokenizer_init, tokenizer_decode, tokenizer_free
#include "tokenizer.h"

// ----------------------------------------------------------------------------
// CUDA precision settings

enum PrecisionMode {
    PRECISION_FP32,
    PRECISION_FP16,
    PRECISION_BF16
};

// Default Properties
typedef float floatN;
#define CUBLAS_LOWP_COMPUTE cublas_compute_type
#ifdef MULTI_GPU
const ncclDataType_t ncclFloatN = ncclFloat;
#endif

// Specific configurations based on the enabled precision
#if defined(ENABLE_FP32)
typedef float floatX;
#define CUBLAS_LOWP HIP_R_32F
#define PRECISION_MODE PRECISION_FP32
const char* load_filename = "gpt2_124M.bin";
const char* precision_mode_str = "fp32";
#ifdef MULTI_GPU
const ncclDataType_t ncclFloatX = ncclFloat;
#endif

// use fp16 (note: this may require gradient scaler, currently not implemented!)
#elif defined(ENABLE_FP16)
typedef half floatX;
#define CUBLAS_LOWP HIP_R_16F
#define PRECISION_MODE PRECISION_FP16
const char* load_filename = "gpt2_124M.bin";
const char* precision_mode_str = "fp16";
#ifdef MULTI_GPU
const ncclDataType_t ncclFloatX = ncclHalf;
#endif

#else // Default to bfloat16
typedef __hip_bfloat16 floatX;
#define CUBLAS_LOWP HIP_R_16BF
#define PRECISION_MODE PRECISION_BF16
const char* load_filename = "gpt2_124M_bf16.bin"; // bf16 weights specific filename
const char* precision_mode_str = "bf16";
#ifdef MULTI_GPU
const ncclDataType_t ncclFloatX = ncclBfloat16;
#endif
#endif

#ifdef ENABLE_CUDNN
#include <cudnn_frontend.h>
namespace fe = cudnn_frontend;
#if CUBLAS_LOWP == HIP_R_16BF
#define CUDNN_16BIT fe::DataType_t::BFLOAT16
#else
#define CUDNN_16BIT fe::DataType_t::HALF
#endif

static hipdnnHandle_t cudnn_handle;
static size_t cudnn_workspace_size = 0; // dynamically allocated as needed (up to 256MiB!)
static void* cudnn_workspace = NULL;
#define checkCudnnErr(err) assert((int)err == 0);
#endif // ENABLE_CUDNN

// ----------------------------------------------------------------------------
// CUDA utils

// cuBLAS workspace. Hardcoding to 32MiB but only Hopper needs 32, for others 4 is OK
static size_t cublaslt_workspace_size = 32 * 1024 * 1024;
static void* cublaslt_workspace = NULL;
static hipblasComputeType_t cublas_compute_type;
hipblasHandle_t cublas_handle;
hipblasLtHandle_t cublaslt_handle;
int cuda_arch_major = 0;
int cuda_arch_minor = 0;
int cuda_num_SMs = 0; // for persistent threads where we want 1 threadblock per SM

namespace cg = cooperative_groups;

// convenience macro for calculating grid/block dimensions for kernels
#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))

// CUDA error checking
void cudaCheck(hipError_t error, const char *file, int line) {
  if (error != hipSuccess) {
    printf("[CUDA ERROR] at file %s:%d:\n%s\n", file, line,
           hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
};
#define cudaCheck(err) (cudaCheck(err, __FILE__, __LINE__))

// cuBLAS error checking
void cublasCheck(hipblasStatus_t status, const char *file, int line)
{
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("[cuBLAS ERROR]: %d %s %d\n", status, file, line);
        exit(EXIT_FAILURE);
    }
}
#define cublasCheck(status) { cublasCheck((status), __FILE__, __LINE__); }

#ifdef MULTI_GPU
void nccl_check(ncclResult_t status, const char *file, int line) {
    if (status != ncclSuccess) {
        printf("[NCCL ERROR] at file %s:%d:\n%s\n", file, line, ncclGetErrorString(status));
        exit(EXIT_FAILURE);
    }
}
#define ncclCheck(err) (nccl_check(err, __FILE__, __LINE__))

void mpi_check(int status, const char *file, int line) {
    if (status != MPI_SUCCESS) {
        char mpi_error[4096];
        int mpi_error_len = 0;
        assert(MPI_Error_string(status, &mpi_error[0], &mpi_error_len) == MPI_SUCCESS);
        printf("[MPI ERROR] at file %s:%d:\n%.*s\n", file, line, mpi_error_len, mpi_error);
        exit(EXIT_FAILURE);
    }
}
#define mpiCheck(err) (mpi_check(err, __FILE__, __LINE__))
#endif

// GPU helper functions for atomicAdd on smaller than 32-bit types
#ifdef ENABLE_BF16
__device__ void atomicAddX(__hip_bfloat16* addr, __hip_bfloat16 val) {
    uintptr_t ptr_val = reinterpret_cast<uintptr_t>(addr);
    __hip_bfloat162* ptr_bf16 = reinterpret_cast<__hip_bfloat162*>(ptr_val & ~uintptr_t(0x3));

    // Prepare the value to add, setting the other half to zero
    __hip_bfloat162 add_val = (ptr_val & 0x3) ? __halves2bfloat162(__ushort_as_bfloat16(0), val)
                                             : __halves2bfloat162(val, __ushort_as_bfloat16(0));
    atomicAdd(ptr_bf16, add_val);
}
#endif

#ifdef ENABLE_FP16
__device__ void atomicAddX(half* addr, half val) {
    uintptr_t ptr_val = reinterpret_cast<uintptr_t>(addr);
    half2* ptr_fp16 = reinterpret_cast<half2*>(ptr_val & ~uintptr_t(0x3));

    // Prepare the value to add, setting the other half to zero
    half2 add_val = (ptr_val & 0x3) ? __halves2half2(__ushort_as_half(0), val)
                                    : __halves2half2(val, __ushort_as_half(0));
    atomicAdd(ptr_fp16, add_val);
}
#endif

__device__ void atomicAddX(float* addr, float val) {
    atomicAdd(addr, val);
}

// ----------------------------------------------------------------------------
// Packed128 data structure, which forces the compiler to use 128-bit loads/stores
// in GPUs that support (the LDG.128 and STS.128 instructions)
// This is a bit similar to the use of float4 in the case of 32-bit floats, but
// supports arbitrary precision.

template<class ElementType>
struct alignas(16) Packed128 {
    __device__ Packed128() = default;
    __device__ explicit Packed128(int4 bits) {
        static_assert(sizeof(bits) == sizeof(payload), "Size mismatch.");
        memcpy(&payload, &bits, sizeof(bits));
    }

    __device__ ElementType& operator[](int index) {
        return payload[index];
    }
    __device__ const ElementType& operator[](int index) const {
        return payload[index];
    }
    __device__ int4 get_bits() const {
        int4 bits;
        static_assert(sizeof(bits) == sizeof(payload), "Size mismatch.");
        memcpy(&bits, &payload, sizeof(bits));
        return bits;
    }

    static constexpr const size_t size = sizeof(int4) / sizeof(ElementType);
    ElementType payload[size];
};

// short-form typedef
typedef Packed128<float> f128;
typedef Packed128<floatX> x128;

// load a Packed128 from an aligned memory address
template<class ElementType>
__device__ Packed128<ElementType> load128(const ElementType* address) {
    return Packed128<ElementType>{*reinterpret_cast<const int4*>(address)};
}

// load a Packed128 from an aligned memory address with streaming cache hint
template<class ElementType>
__device__ Packed128<ElementType> load128cs(const ElementType* address) {
    return Packed128<ElementType>{__ldcs(reinterpret_cast<const int4*>(address))};
}

// store a Packed128 to an aligned memory address
template<class ElementType>
__device__ void store128(ElementType* target, Packed128<ElementType> value) {
    *reinterpret_cast<int4*>(target) = value.get_bits();
}

// store a Packed128 to an aligned memory address with streaming cache hint
template<class ElementType>
__device__ void store128cs(ElementType* target, Packed128<ElementType> value) {
    __stcs(reinterpret_cast<int4*>(target), value.get_bits());
}

// ----------------------------------------------------------------------------
// Random Number Generatiom

// Simple xorshift RNG
__device__ __host__ unsigned int random_u32(unsigned long long *state) {
    // xorshift rng: https://en.wikipedia.org/wiki/Xorshift#xorshift.2A
    *state ^= *state >> 12;
    *state ^= *state << 25;
    *state ^= *state >> 27;
    return (*state * 0x2545F4914F6CDD1Dull) >> 32;
}
__device__ __host__ float random_f32(unsigned long long *state) { // random float32 in [0,1)
    return (random_u32(state) >> 8) / 16777216.0f;
}

// SquirrelNoise5 - Squirrel's Raw Noise utilities (version 5)
// This gives us a random number from threadIdx/blockIdx + a single seed for the entire GPU
// todo - possibly overkill and we don't need such high quality random numbers? (tbd)
// http://eiserloh.net/noise/SquirrelNoise5.hpp
__device__ __host__ constexpr unsigned int SquirrelNoise5(int positionX, unsigned int seed)
{
	constexpr unsigned int SQ5_BIT_NOISE1 = 0xd2a80a3f;	// 11010010101010000000101000111111
	constexpr unsigned int SQ5_BIT_NOISE2 = 0xa884f197;	// 10101000100001001111000110010111
	constexpr unsigned int SQ5_BIT_NOISE3 = 0x6C736F4B; // 01101100011100110110111101001011
	constexpr unsigned int SQ5_BIT_NOISE4 = 0xB79F3ABB;	// 10110111100111110011101010111011
	constexpr unsigned int SQ5_BIT_NOISE5 = 0x1b56c4f5;	// 00011011010101101100010011110101
	unsigned int mangledBits = (unsigned int) positionX;
	mangledBits *= SQ5_BIT_NOISE1;
	mangledBits += seed;
	mangledBits ^= (mangledBits >> 9);
	mangledBits += SQ5_BIT_NOISE2;
	mangledBits ^= (mangledBits >> 11);
	mangledBits *= SQ5_BIT_NOISE3;
	mangledBits ^= (mangledBits >> 13);
	mangledBits += SQ5_BIT_NOISE4;
	mangledBits ^= (mangledBits >> 15);
	mangledBits *= SQ5_BIT_NOISE5;
	mangledBits ^= (mangledBits >> 17);
	return mangledBits;
}
__device__ __host__ constexpr unsigned int Get1dNoiseUint(int positionX, unsigned int seed)
{
	return SquirrelNoise5(positionX, seed);
}
__device__ __host__ constexpr unsigned int Get2dNoiseUint(int indexX, int indexY, unsigned int seed)
{
	constexpr int PRIME_NUMBER = 198491317; // Large prime number with non-boring bits
	return SquirrelNoise5(indexX + (PRIME_NUMBER * indexY), seed);
}
__device__ __host__ constexpr float Get1dNoiseZeroToOne(int index, unsigned int seed)
{
	constexpr double ONE_OVER_MAX_UINT = (1.0 / (double) 0xFFFFFFFF);
	return (float)(ONE_OVER_MAX_UINT * (double) SquirrelNoise5(index, seed));
}
__device__ __host__ constexpr float Get2dNoiseZeroToOne(int indexX, int indexY, unsigned int seed)
{
	constexpr double ONE_OVER_MAX_UINT = (1.0 / (double) 0xFFFFFFFF);
	return (float)(ONE_OVER_MAX_UINT * (double) Get2dNoiseUint(indexX, indexY, seed));
}

// stochastic rounding built on top of Squirel Noise above (with seed updated per step via xorshift)
__device__ __forceinline__ void stochastic_rounding(float in, __hip_bfloat16 *out, unsigned int seed) {
    // todo - is this stochastic rounding *too good*? can we cut any corners?
    unsigned int random = Get2dNoiseUint(threadIdx.x, blockIdx.x, seed);
    unsigned int threshold = random & 0xFFFF;
    unsigned int float_bits = __float_as_uint(in);
    unsigned int rounded_bits = float_bits & 0x0000FFFF;
    float_bits = (rounded_bits > threshold) ? (float_bits | 0xFFFF) : (float_bits  & ~0xFFFF);
    *out = __float2bfloat16_rn(__uint_as_float(float_bits));
}
__device__ __forceinline__ void stochastic_rounding(float in, half *out, unsigned int random) {
    *out = (float)in; // todo - implement this...
}
__device__ __forceinline__ void stochastic_rounding(float in, float *out, unsigned int random) {
    *out = in; // dummy function for when floatX is float (FP32 mode)
}

// ----------------------------------------------------------------------------
// MPI / multi-processing setup

// Parameters specific to training on multiple GPUs.
typedef struct {
    int process_rank;      // Rank of this process among all MPI processes. 0 if no multi-GPU.
    int num_processes;     // Total number of processes. 1 if no multi-GPU.
    int local_device_idx;  // This process GPU index on current machine. 0 if no multi-GPU.
#ifdef MULTI_GPU
    ncclComm_t nccl_comm;  // NCCL communication primitive, used for collective multi-GPU work.
#endif
} MultiGpuConfig;

// one global variable to hold the multi-GPU configuration for this process
MultiGpuConfig multi_gpu_config;

#ifdef MULTI_GPU
// Determine which GPU this process should use.
// Processes on the same machines use different GPU indicies. Processes on other machines don't.
// Copied from NCCL examples: https://docs.nvidia.com/deeplearning/nccl/user-guide/docs/examples.html#example-2-one-device-per-process-or-thread
int multi_gpu_get_local_device_idx(int process_rank, int num_processes) {
  char hostname[1024];
  hostname[1023] = '\0';
  // All processes on the same machine will share the same hostname.
  gethostname(hostname, 1023);
  for (int i=0; i < 1024; i++) {
    if (hostname[i] == '.') {
        hostname[i] = '\0';
        break;
    }
  }
  uint64_t hostname_hash = 5381;
  for (int c = 0; hostname[c] != '\0'; c++){ hostname_hash = ((hostname_hash << 5) + hostname_hash) ^ hostname[c]; }

  // Distribute all hostname hashes to all processes.
  uint64_t* all_hostsname_hashes = (uint64_t*)malloc(num_processes * sizeof(uint64_t));
  all_hostsname_hashes[process_rank] = hostname_hash;
  mpiCheck(MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, all_hostsname_hashes, sizeof(uint64_t), MPI_BYTE, MPI_COMM_WORLD));

  // Identify which GPU we need to use.
  int local_device_idx = 0;
  for (int current_process = 0; current_process < num_processes; ++current_process) {
     if (current_process == process_rank) {
      // Found my gpu, local_device_idx now has my target GPU index.
      break;
     }
     if (all_hostsname_hashes[current_process] == all_hostsname_hashes[process_rank]) {
      // This process ID runs on the same machine, but it's not me, skip this GPU
      local_device_idx++;
     }
  }

  free(all_hostsname_hashes);
  return local_device_idx;
}
#endif

MultiGpuConfig multi_gpu_config_init(int *argc, char ***argv) {
#ifdef MULTI_GPU
    // Initialize MPI.
    MultiGpuConfig result;
    mpiCheck(MPI_Init(argc, argv));
    mpiCheck(MPI_Comm_rank(MPI_COMM_WORLD, &result.process_rank));
    mpiCheck(MPI_Comm_size(MPI_COMM_WORLD, &result.num_processes));
    result.local_device_idx = multi_gpu_get_local_device_idx(result.process_rank, result.num_processes);
    cudaCheck(hipSetDevice(result.local_device_idx));
    ncclUniqueId nccl_id;
    if (result.process_rank == 0) {
        ncclCheck(ncclGetUniqueId(&nccl_id));
    }
    mpiCheck(MPI_Bcast((void *)&nccl_id, sizeof(nccl_id), MPI_BYTE, 0, MPI_COMM_WORLD));
    ncclCheck(ncclCommInitRank(&result.nccl_comm, result.num_processes, nccl_id, result.process_rank));
    return result;
#else
    printf("Multi-GPU support is disabled. Using a single GPU.\n");
    MultiGpuConfig result;
    result.process_rank = 0;
    result.num_processes = 1;
    result.local_device_idx = 0;
    return result;
#endif
}

void multi_gpu_config_free(const MultiGpuConfig* multi_gpu_config) {
#ifdef MULTI_GPU
    ncclCheck(ncclCommDestroy(multi_gpu_config->nccl_comm));
    mpiCheck(MPI_Finalize());
#endif
}

// convenience function that only prints if the rank of process is zero
void printf0(const char *format, ...) {
    if (multi_gpu_config.process_rank == 0) {
        va_list args;
        va_start(args, format);
        vprintf(format, args);
        va_end(args);
    }
}

// ----------------------------------------------------------------------------
// cuDNN path
#ifdef ENABLE_CUDNN

using graph_tensors_fwd = std::tuple<std::shared_ptr<fe::graph::Graph>,
                                     std::shared_ptr<fe::graph::Tensor_attributes>,  // Q,
                                     std::shared_ptr<fe::graph::Tensor_attributes>,  // K,
                                     std::shared_ptr<fe::graph::Tensor_attributes>,  // V,
                                     std::shared_ptr<fe::graph::Tensor_attributes>,  // Attn_scale,
                                     std::shared_ptr<fe::graph::Tensor_attributes>,  // O
                                     std::shared_ptr<fe::graph::Tensor_attributes>>; // Stats

using graph_tensors_bwd = std::tuple<std::shared_ptr<fe::graph::Graph>,
                                     std::shared_ptr<fe::graph::Tensor_attributes>,  // Q,
                                     std::shared_ptr<fe::graph::Tensor_attributes>,  // K,
                                     std::shared_ptr<fe::graph::Tensor_attributes>,  // V,
                                     std::shared_ptr<fe::graph::Tensor_attributes>,  // O
                                     std::shared_ptr<fe::graph::Tensor_attributes>,  // dO
                                     std::shared_ptr<fe::graph::Tensor_attributes>,  // Stats
                                     std::shared_ptr<fe::graph::Tensor_attributes>,  // Attn_scale,
                                     std::shared_ptr<fe::graph::Tensor_attributes>,  // dQ,
                                     std::shared_ptr<fe::graph::Tensor_attributes>,  // dK,
                                     std::shared_ptr<fe::graph::Tensor_attributes>>; // dV

// Need a cache because graph->build_operation_graph() is slow but everything else seems fast
using cache_type_fwd = std::unordered_map<std::size_t, graph_tensors_fwd>;
using cache_type_bwd = std::unordered_map<std::size_t, graph_tensors_bwd>;

// Loosely based on cuDNN frontend samples functions and massively simplified
template <typename... Args>
auto lookup_cache_or_build_graph_fwd(Args... args) {
    static cache_type_fwd user_maintained_cache_fwd;
    auto [B, H, T, HS, is_inference_only] = std::make_tuple(args...);

    auto graph = std::make_shared<fe::graph::Graph>();
    graph->set_io_data_type(CUDNN_16BIT)
          .set_intermediate_data_type(fe::DataType_t::FLOAT)
          .set_compute_data_type(fe::DataType_t::FLOAT);

    // QKV is (B, T, 3, NH, HS) which cuDNN can handle directly without an external permute
    auto Q = graph->tensor(fe::graph::Tensor_attributes()
                               .set_name("Q")
                               .set_dim({B, H, T, HS})
                               .set_stride({3 * H * HS * T,  HS, 3 * H * HS, 1}));
    auto K = graph->tensor(fe::graph::Tensor_attributes()
                               .set_name("K")
                               .set_dim({B, H, T, HS})
                               .set_stride({3 * H * HS * T, HS, 3 * H * HS, 1}));
    auto V = graph->tensor(fe::graph::Tensor_attributes()
                               .set_name("V")
                               .set_dim({B, H, T, HS})
                               .set_stride({3 * H * HS * T, HS, 3 * H * HS, 1}));
    auto attn_scale = graph->tensor(fe::graph::Tensor_attributes()
                                .set_name("attn_scale")
                                .set_dim({1, 1, 1, 1})
                                .set_stride({1, 1, 1, 1})
                                .set_is_pass_by_value(true)
                                .set_data_type(fe::DataType_t::FLOAT));

    auto sdpa_options = fe::graph::SDPA_attributes().set_name("flash_attention");
    sdpa_options.set_is_inference(is_inference_only);
    sdpa_options.set_attn_scale(attn_scale);
    sdpa_options.set_causal_mask(true);

    // Create the graph operation and get the output tensors back
    auto [O, stats] = graph->sdpa(Q, K, V, sdpa_options);

    // Output is (B, T, NH, HS) BF16/FP16 and stats for backward pass is (B, NH, T) FP32
    O->set_output(true).set_dim({B, H, T, HS}).set_stride({H * HS * T, HS, H * HS, 1});

    assert(stats == nullptr || is_inference_only == false);
    if (is_inference_only == false) {
        stats->set_output(true).set_data_type(fe::DataType_t::FLOAT)
                               .set_dim({B, H, T, 1})
                               .set_stride({H * T, T, 1, 1});
    }

    assert(graph->validate().is_good());
    auto key = graph->key();
    auto it = user_maintained_cache_fwd.find(key);
    if (it != user_maintained_cache_fwd.end()) {
        return it->second;
    }

    // Build the operation graph and execution part (this is the VERY SLOW PART)
    assert(graph->build_operation_graph(cudnn_handle).is_good());
    auto plans = graph->create_execution_plans({fe::HeurMode_t::A});
    assert(graph->check_support(cudnn_handle).is_good());
    assert(graph->build_plans(cudnn_handle).is_good());

    auto tuple = std::make_tuple(graph, Q, K, V, attn_scale, O, stats);
    user_maintained_cache_fwd.insert({key, tuple});
    return tuple;
}

template <typename... Args>
auto lookup_cache_or_build_graph_bwd(Args... args) {
    static cache_type_bwd user_maintained_cache_bwd;
    auto [B, NH, T, HS] = std::make_tuple(args...);

    auto graph = std::make_shared<fe::graph::Graph>();
    graph->set_io_data_type(CUDNN_16BIT)
          .set_intermediate_data_type(fe::DataType_t::FLOAT)
          .set_compute_data_type(fe::DataType_t::FLOAT);

    // (B, N, 3, NH, HS)
    // must come from inp (which means we also need to convert THAT to FP16)
    auto Q = graph->tensor(fe::graph::Tensor_attributes()
                            .set_name("Q")
                            .set_dim({B, NH, T, HS})
                            .set_stride({3 * NH * HS * T, HS, 3 * NH * HS, 1}));
    auto K = graph->tensor(fe::graph::Tensor_attributes()
                            .set_name("K")
                            .set_dim({B, NH, T, HS})
                            .set_stride({3 * NH * HS * T, HS, 3 * NH * HS, 1}));
    auto V = graph->tensor(fe::graph::Tensor_attributes()
                            .set_name("V")
                            .set_dim({B, NH, T, HS})
                            .set_stride({3 * NH * HS * T, HS, 3 * NH * HS, 1}));
    auto O = graph->tensor(fe::graph::Tensor_attributes()
                            .set_name("O")
                            .set_dim({B, NH, T, HS})
                            .set_stride({NH * HS * T, HS, NH * HS, 1}));
    auto dO = graph->tensor(fe::graph::Tensor_attributes()
                            .set_name("dO")
                            .set_dim({B, NH, T, HS})
                            .set_stride({NH * HS * T, HS, NH * HS, 1}));

    auto stats = graph->tensor(fe::graph::Tensor_attributes()
                            .set_name("stats")
                            .set_dim({B, NH, T, 1})
                            .set_stride({NH * T, T, 1, 1})
                            .set_data_type(fe::DataType_t::FLOAT));
    auto attn_scale = graph->tensor(fe::graph::Tensor_attributes()
                            .set_name("attn_scale")
                            .set_dim({1, 1, 1, 1})
                            .set_stride({1, 1, 1, 1})
                            .set_is_pass_by_value(true)
                            .set_data_type(fe::DataType_t::FLOAT));
    auto sdpa_backward_options = fe::graph::SDPA_backward_attributes()
                            .set_name("flash_attention_backward")
                            .set_causal_mask(true)
                            .set_attn_scale(attn_scale);

    // Create the graph operation and get the output tensors back
    auto [dQ, dK, dV] = graph->sdpa_backward(Q, K, V, O, dO, stats, sdpa_backward_options);

    dQ->set_output(true).set_dim({B, NH, T, HS}).set_stride({3 * NH * HS * T, HS, 3 * NH * HS, 1});
    dK->set_output(true).set_dim({B, NH, T, HS}).set_stride({3 * NH * HS * T, HS, 3 * NH * HS, 1});
    dV->set_output(true).set_dim({B, NH, T, HS}).set_stride({3 * NH * HS * T, HS, 3 * NH * HS, 1});

    assert(graph->validate().is_good());
    auto key = graph->key();
    auto it = user_maintained_cache_bwd.find(key);
    if (it != user_maintained_cache_bwd.end()) {
        return it->second;
    }

    // Build the operation graph and execution part (this is the VERY SLOW PART)
    assert(graph->build_operation_graph(cudnn_handle).is_good());
    auto plans = graph->create_execution_plans({fe::HeurMode_t::A});
    assert(graph->check_support(cudnn_handle).is_good());
    assert(graph->build_plans(cudnn_handle).is_good());

    auto tuple = std::make_tuple(graph, Q, K, V, O, dO, stats, attn_scale, dQ, dK, dV);
    user_maintained_cache_bwd.insert({key, tuple});
    return tuple;
}

void attention_forward_cudnn(floatX* out,  // output: (B, T, NH, HS)
                             float* stats, // output for backward pass: (B, NH, T)
                             floatX* inp,  // input: (B, T, 3, NH, HS) QKV
                             int B, int T, int NH, int C) {
    int HS = C / NH; // number of features per head
    bool is_inference_only = (stats == nullptr);

    // Get graph and tensors from cache (or generate it on first use)
    auto [graph, Q, K, V, attn_scale, O, softmax_stats] =
        lookup_cache_or_build_graph_fwd(B, NH, T, HS, is_inference_only);

    // Prepare all the tensor pointers for executing the graph
    void* devPtrQ = inp;
    void* devPtrK = (inp + C);
    void* devPtrV = (inp + 2 * C);
    float attn_scale_cpu = 1.0 / sqrtf(HS);
    void* devPtrO = out;

    // Build variant pack
    std::unordered_map<std::shared_ptr<fe::graph::Tensor_attributes>, void*> variant_pack = {
        {Q, devPtrQ}, {K, devPtrK}, {V, devPtrV}, {attn_scale, &attn_scale_cpu}, {O, devPtrO}};

    // Add the stats tensor unless we are only doing inference (only needed for backward pass)
    if (is_inference_only == false) {
        variant_pack[softmax_stats] = stats;
    }

    // Reallocate the workspace if the required size is greater than the current workspace
    // By default, cuDNN uses up to 256MiB of workspace, so we don't want to just allocate the maximum
    if (graph->get_workspace_size() > cudnn_workspace_size) {
        if (cudnn_workspace_size > 0) {
            cudaCheck(hipFree(cudnn_workspace));
        }
        cudnn_workspace_size = graph->get_workspace_size();
        cudaCheck(hipMalloc(&cudnn_workspace, cudnn_workspace_size));
    }

    // Execute graph
    assert(graph->execute(cudnn_handle, variant_pack, cudnn_workspace).is_good());
    cudaCheck(hipGetLastError());
}

void attention_backward_cudnn(floatX* dqkvr,                                       // output
                              floatX* dout, floatX* qkvr, floatX* o, float* stats, // inputs
                              int B, int T, int NH, int C) {
    int HS = C / NH; // number of features per head

    // Get graph and tensors from cache (or generate it on first use)
    auto [graph, Q, K, V, O, dO, Stats, attn_scale, dQ, dK, dV] =
        lookup_cache_or_build_graph_bwd(B, NH, T, HS);

    // Prepare all the tensor pointers for executing the graph
    void* devPtrQ = qkvr;
    void* devPtrK = (qkvr + NH * HS);
    void* devPtrV = (qkvr + 2 * NH * HS);
    void* devPtrO = o;
    void* devPtrdO = dout;
    void* devPtrStats = stats;
    float attn_scale_cpu = 1.0 / sqrtf(HS);

    void* devPtrdQ = dqkvr;
    void* devPtrdK = (dqkvr + NH * HS);
    void* devPtrdV = (dqkvr + 2 * NH * HS);

    // Build variant pack that links each tensor to its data pointer
    std::unordered_map<std::shared_ptr<fe::graph::Tensor_attributes>, void*> variant_pack = {
        {Q, devPtrQ}, {K, devPtrK}, {V, devPtrV}, {O, devPtrO}, {dO, devPtrdO}, {Stats, devPtrStats},
        {dQ, devPtrdQ}, {dK, devPtrdK}, {dV, devPtrdV},
        {attn_scale, &attn_scale_cpu}};

    // Reallocate the workspace if the required size is greater than the current workspace
    // By default, cuDNN uses up to 256MiB of workspace, so we don't want to just allocate the maximum
    if (graph->get_workspace_size() > cudnn_workspace_size) {
        if (cudnn_workspace_size > 0) {
            cudaCheck(hipFree(cudnn_workspace));
        }
        cudnn_workspace_size = graph->get_workspace_size();
        cudaCheck(hipMalloc(&cudnn_workspace, cudnn_workspace_size));
    }

    // Execute graph
    assert(graph->execute(cudnn_handle, variant_pack, cudnn_workspace).is_good());
    cudaCheck(hipGetLastError());
}
#endif // ENABLE_CUDNN

// ----------------------------------------------------------------------------
// all the kernels

__global__ void encoder_forward_kernel2(floatX* out,
                               int* inp, floatX* wte, floatX* wpe,
                               int B, int T, int C) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int N = B * T * C;

    if (idx < N) {
        int bt = idx / C;
        int b = bt / T;
        int t = bt % T;
        int c = idx % C;

        int ix = inp[b * T + t];

        floatX* out_btc = out + b * T * C + t * C + c;
        floatX* wte_ix = wte + ix * C + c;
        floatX* wpe_tc = wpe + t * C + c;
        *out_btc = (floatX)((float)*wte_ix + (float)*wpe_tc);
    }
}

// really bad naive kernel with atomicAdd
__global__ void encoder_backward_kernel(floatX* dwte, floatX* dwpe,
                                        const floatX* dout, const int* inp,
                                        int B, int T, int C) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int N = B * T * C;

    if (idx < N) {
        int bt = idx / C;
        int b = bt / T;
        int t = bt % T;
        int c = idx % C;

        int ix = inp[b * T + t];

        const floatX* dout_btc = dout + b * T * C + t * C + c;
        floatX* dwte_ix = dwte + ix * C + c;
        floatX* dwpe_tc = dwpe + t * C + c;

        atomicAddX(dwte_ix, (floatX)*dout_btc);
        atomicAddX(dwpe_tc, (floatX)*dout_btc);
    }
}

__global__ void layernorm_forward_kernel3(floatX* __restrict__ out, floatX* __restrict__ mean, floatX* __restrict__ rstd,
                                    const floatX*  __restrict__ inp, const floatX*  __restrict__ weight,
                                    const floatX* __restrict__ bias, int N, int C) {
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    int idx = blockIdx.x * warp.meta_group_size() + warp.meta_group_rank();
    if(idx >= N) { return; } // guard

    // the row of input that this group of threads is responsible for
    const floatX* x = inp + idx * C;

    // mean
    float sum = 0.0f;
    for (int i = warp.thread_rank(); i < C; i += warp.size()) {
        sum += (float)x[i];
    }
    sum = cg::reduce(warp, sum, cg::plus<float>{});
    float m = sum / C;
    if(warp.thread_rank() == 0 && mean != nullptr) {
        __stcs(mean + idx, (floatX)m);
    }

    // rstd
    sum = 0.0f;
    for (int i = warp.thread_rank(); i < C; i += warp.size()) {
        float diff = (float)x[i] - m;
        sum += diff * diff;
    }
    sum = cg::reduce(warp, sum, cg::plus<float>{});
    float s = rsqrtf(sum / C + 1e-5f);
    if(warp.thread_rank() == 0 && rstd != nullptr) {
        __stcs(rstd + idx, (floatX)s);
    }

    // final normalization and scaling by weight/bias
    floatX* o = out + idx * C;
    for (int c = warp.thread_rank(); c < C; c += warp.size()) {
        // load and store using the .cs "streaming" hint to the compiler,
        // indicating that this data will not be reused soon, and can be streamed through the caches
        // this allows the threads to get more cache-hits for the (shared) weight and bias parameters
        float n = s * ((float)__ldcs(x+c) - m);
        __stcs(o+c, (floatX)(n * (float)weight[c] + (float)bias[c]));
    }
}

// inputs floatX, outputs FP32 (for current FP32-only activation path for this WIP)
__global__ void permute_kernel(floatX* q, floatX* k, floatX* v,
                               const floatX* inp,
                               int B, int N, int NH, int d) {
    // okay so now, this kernel wants Q,K,V to all be of shape (B, NH, N, d)
    // but instead, we have a single tensor QKV (inp) of shape (B, N, 3, NH, d)
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Q[b][nh_][n][d_] = inp[b][n][0][nh_][d_]
    if (idx < B * NH * N * d) {
        int b = idx / (NH * N * d);
        int rest = idx % (NH * N * d);
        int nh_ = rest / (N * d);
        rest = rest % (N * d);
        int n = rest / d;
        int d_ = rest % d;
        int inp_idx = (b * N * 3 * NH * d) + (n * 3 * NH * d) + (0 * NH * d) + (nh_ * d) + d_;
        q[idx] = __ldcs(&inp[inp_idx]);
        k[idx] = __ldcs(&inp[inp_idx + NH * d]);
        v[idx] = __ldcs(&inp[inp_idx + 2 * (NH * d)]);
    }
}

__global__ void permute_kernel_backward(floatX* dinp,
                                        const floatX* dq, const floatX* dk, const floatX* dv,
                                        int B, int N, int NH, int d) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < B * NH * N * d) {
        int b = idx / (NH * N * d);
        int rest = idx % (NH * N * d);
        int nh_ = rest / (N * d);
        rest = rest % (N * d);
        int n = rest / d;
        int d_ = rest % d;

        int inp_idx = (b * N * 3 * NH * d) + (n * 3 * NH * d) + (0 * NH * d) + (nh_ * d) + d_;
        dinp[inp_idx] = dq[idx];
        dinp[inp_idx + NH * d] = dk[idx];
        dinp[inp_idx + 2 * (NH * d)] = dv[idx];
    }
}

__global__ void unpermute_kernel(floatX* inp, floatX *out, int B, int N, int NH, int d) {
   // out has shape (B, nh, N, d) but we need to unpermute it to (B, N, nh, d)

    int idx = (blockIdx.x * blockDim.x + threadIdx.x);
    // out[b][n][nh_][d_] <- inp[b][nh_][n][d_]
    if (idx < B * NH * N * d) {
        int b = idx / (NH * N * d);
        int rest = idx % (NH * N * d);
        int nh_ = rest / (N * d);
        rest = rest % (N * d);
        int n = rest / d;
        int d_ = rest % d;
        int other_idx = (b * NH * N * d) + (n * NH * d) + (nh_ * d) + d_;
        out[other_idx] = __ldcs(&inp[idx]);
    }
}

__global__ void unpermute_kernel_backward(floatX* dinp, const floatX *dout, int B, int N, int NH, int d) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < B * NH * N * d) {
        int b = idx / (NH * N * d);
        int rest = idx % (NH * N * d);
        int nh_ = rest / (N * d);
        rest = rest % (N * d);
        int n = rest / d;
        int d_ = rest % d;
        int other_idx = (b * NH * N * d) + (n * NH * d) + (nh_ * d) + d_;
        dinp[idx] = (floatX)dout[other_idx];
    }
}

__global__ void softmax_forward_kernel5(floatX* out, float inv_temperature, const floatX* inp, int N, int T) {
    // inp, out shape: (N, T, T), where N = B * NH
    // fuses the multiplication by scale inside attention
    // directly autoregressive, so we only compute the lower triangular part
    // uses the online softmax algorithm
    assert(T % 4  == 0);
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    // micro-optimization: we iterate backwards so that
    // after the softmax backward operation completes, the cache retains the
    // part of the matrix close to the upper left corner, which benefits the
    // matmul operation that immediately follows.
    // int idx = blockIdx.x * warp.meta_group_size() + warp.meta_group_rank(); // forward order
    int idx = (gridDim.x - blockIdx.x - 1) * warp.meta_group_size() + warp.meta_group_rank(); // backward order
    if(idx >= N * T) {
        return;
    }
    int own_pos = idx % T;
    int pos_by_4 = own_pos / 4;

    // one row of inp, i.e. inp[idx, :] of shape (T,)
    const floatX* x = inp + idx * T;

    // not INF, so we don't get NaNs accidentally when subtracting two values.
    float maxval = -FLT_MAX;
    float sumval = 0.0f;

    const floatX* x_aligned = reinterpret_cast<const floatX*>(__builtin_assume_aligned(x, 16));
    for (int i = warp.thread_rank(); i < pos_by_4; i += warp.size()) {
        float regarray[4];
        #pragma unroll
        for (int k = 0; k < 4; ++k) {
            regarray[k] = (float)x_aligned[4*i + k];
        }
        float old_maxval = maxval;
        for(int k = 0; k < 4; ++k) {
            maxval = fmaxf(maxval, regarray[k]);
        }
        sumval *= expf(inv_temperature * (old_maxval - maxval));
        for(int k = 0; k < 4; ++k) {
            sumval += expf(inv_temperature * (regarray[k] - maxval));
        }
    }

    if(4*pos_by_4 + warp.thread_rank() <= own_pos) {
        float old_maxval = maxval;
        maxval = fmaxf(maxval, (float)x[4*pos_by_4 + warp.thread_rank()]);
        sumval *= expf(inv_temperature * (old_maxval - maxval));
        sumval += expf(inv_temperature * ((float)x[4*pos_by_4 + warp.thread_rank()] - maxval));
    }

    float global_maxval = cg::reduce(warp, maxval, cg::greater<float>{});
    sumval *= expf(inv_temperature * (maxval - global_maxval));

    float sum = cg::reduce(warp, sumval, cg::plus<float>{});
    float norm = 1.f / sum;

    // divide the whole row by the sum
    for (int i = warp.thread_rank(); i <= own_pos; i += warp.size()) {
        // recalculation is faster than doing the round-trip through memory.
        float ev = expf(inv_temperature * ((float)__ldcs(x + i) - global_maxval));
        __stcs(out + idx * T + i, (floatX)(ev * norm));
    }
}

__global__ void residual_forward_kernel(floatX* out, floatX* inp1, floatX* inp2, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        out[idx] = (floatX)((float)__ldcs(&inp1[idx]) + (float)__ldcs(&inp2[idx]));
    }
}

#define GELU_SCALING_FACTOR sqrtf(2.0f / M_PI)
__global__ void gelu_forward_kernel2(floatX* out, const floatX* inp, int N) {
    int i = (blockIdx.x * blockDim.x + threadIdx.x) * x128::size;
    if (i < N) {
        x128 packed_out;
        x128 packed_inp = load128cs(inp + i); // load and do not keep in cache
        for(int k = 0; k < packed_inp.size; ++k) {
            float xi = (float)packed_inp[k];
            float cube = 0.044715f * xi * xi * xi;
            packed_out[k] = (floatX)(0.5f * xi * (1.0f + tanhf(GELU_SCALING_FACTOR * (xi + cube))));
        }
        // store instead of storecs (without cache streaming) in case it is useful for the
        // data to be in the cache for the next operation after this GeLU
        store128(out + i, packed_out);
    }
}

__global__ void gelu_backward_kernel(floatX* dinp, const floatX* inp, const floatX* dout, const int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        float x = (float)inp[i];
        float cube = 0.044715f * x * x * x;
        float tanh_arg = GELU_SCALING_FACTOR * (x + cube);
        float tanh_out = tanhf(tanh_arg);
        float coshf_out = coshf(tanh_arg);
        float sech_out = 1.0f / (coshf_out * coshf_out);
        float local_grad = 0.5f * (1.0f + tanh_out) + x * 0.5f * sech_out * GELU_SCALING_FACTOR * (1.0f + 3.0f * 0.044715f * x * x);
        dinp[i] = (floatX)(local_grad * (float)dout[i]);
    }
}

// this kernel performs a column-wise reduction over dout, in PyTorch equivalent to:
// dbias = dout.sum((0,1))
// the idea is to employ one block to reduce along several columns,
// where each block has a width of 32 columns to ensure coalesced access.
// at the end we accumulate the reductions performed by the warps in each block via shared memory
__global__ void matmul_backward_bias_kernel4(floatX* dbias, const floatX* dout, int B, int T, int OC) {
    // this kernel is launched with 1D grid_dim of OC/32
    // for example let's say block_size is 128
    extern __shared__ float smem[]; // of size block_size (128)
    const int warp_id = threadIdx.x / warpSize; // warp index in the block, 0,1,2,3
    const int lane_id = threadIdx.x % warpSize; // thread index in the warp, 0,1,2,...,31
    const int tl = blockIdx.x * warpSize; // pointer to the start column for this block
    const int vstep = blockDim.x / warpSize; // number of warps in a block, e.g. 4

    // pointer to the start of the column for one lane of threads
    // so e.g. 4 threads (of the same lane_id) will reduce this one column
    const floatX* dout_col = dout + tl + lane_id;

    // column reductions by looping through the rows
    // each of the 4 threads offsets by its warp_id and then skips by vstep
    // together these 4 threads cover all B*T rows of this (lane_id) column
    // importantly, consecutive threads (in threadId) are processing adjacent columns,
    // leading to a coalesced memory access pattern
    float dout_sum = 0.0f;
    for (int row = warp_id; row < B * T; row += vstep) {
        dout_sum += (float)dout_col[row * OC];
    }
    smem[lane_id + warp_id * warpSize] = dout_sum;
    __syncthreads();

    // warp_id 0 reduces the shared memory column-wise, linearly
    dout_sum = 0.0f;
    if (warp_id == 0) {
        for (int j = 0; j < vstep; j++) {
            dout_sum += smem[lane_id + j * warpSize];
        }
        dbias[tl + lane_id] = (floatX)dout_sum;
    }
}

// single FP32 scratchpad shared by all the threadblocks (based on kernels 3 & 5)
__global__ void layernorm_backward_kernel6(floatX* dinp, floatX* dweight, floatX* dbias, float* scratch,
                        const floatX* dout, const floatX* inp, const floatX* weight, const floatX* mean, const floatX* rstd,
                        int B, int T, int C) {
    extern __shared__ float shared[]; // size = 2 * C + 1

    namespace cg = cooperative_groups;
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    int base_idx = blockIdx.x * warp.meta_group_size() + warp.meta_group_rank();


    // the first half of shared memory is bias, second is weight
    float* dbias_shared = shared;
    float* dweight_shared = shared + C;

    // init shared memory to zero
    #pragma unroll 4
    for(int i = threadIdx.x; i < C; i+= blockDim.x){
       dbias_shared[i] = 0.0f;
       dweight_shared[i] = 0.0f;
    }
    unsigned int *tmp_flag = (unsigned int*)(shared + C*2);
    __syncthreads();

    int warps_in_grid = gridDim.x * warp.meta_group_size();
    for (int idx = base_idx; idx < B * T; idx += warps_in_grid) {
        int b = idx / T;
        int t = idx % T;

        const floatX* dout_bt = dout + b * T * C + t * C;
        const floatX* inp_bt = inp + b * T * C + t * C;
        floatX* dinp_bt = dinp + b * T * C + t * C;
        const float mean_bt = (float)mean[b * T + t];
        const float rstd_bt = (float)rstd[b * T + t];

        // first: two reduce operations
        float dnorm_mean = 0.0f;
        float dnorm_norm_mean = 0.0f;
        for (int i = warp.thread_rank(); i < C; i  += warp.size()) {
            float norm_bti = ((float)inp_bt[i] - mean_bt) * rstd_bt;
            float dnorm_i = (float)weight[i] * (float)dout_bt[i];
            dnorm_mean += dnorm_i;
            dnorm_norm_mean += dnorm_i * norm_bti;
        }
        dnorm_mean = cg::reduce(warp, dnorm_mean, cg::plus<float>{});
        dnorm_norm_mean = cg::reduce(warp, dnorm_norm_mean, cg::plus<float>{});
        dnorm_mean = dnorm_mean / C;
        dnorm_norm_mean = dnorm_norm_mean / C;

        // now iterate again and accumulate all the gradients
        for (int i = warp.thread_rank(); i < C; i += warp.size()) {
            float dout_i = (float)__ldcs(&dout_bt[i]);
            float norm_bti = ((float)__ldcs(&inp_bt[i]) - mean_bt) * rstd_bt;
            float dnorm_i = (float)weight[i] * dout_i;
            // gradient contribution to bias
            atomicAdd(&dbias_shared[i], dout_i);
            // gradient contribution to weight
            atomicAdd(&dweight_shared[i], norm_bti * dout_i);
            // gradient contribution to input
            float dval = 0.0f;
            dval += dnorm_i; // term 1
            dval -= dnorm_mean; // term 2
            dval -= norm_bti * dnorm_norm_mean; // term 3
            dval *= rstd_bt; // final scale
            dinp_bt[i] = (floatX)((float)dinp_bt[i] + dval);
        }
    }

    // Accumulate into a FP32 scratchpad
    // BF16 atomics are potentially much slower... and this is more precise!
    // todo - could potentially avoid the extra copy if floatX is FP32, fairly negligible though
    __syncthreads();
    float* scratch_dbias = scratch;
    float* scratch_dweight = scratch + C;
    unsigned int* scratchFlag = (unsigned int*)(scratch + (2 * C));
    for(int i = threadIdx.x; i < C; i+= blockDim.x) {
        atomicAdd(&scratch_dbias[i], dbias_shared[i]);
        atomicAdd(&scratch_dweight[i], dweight_shared[i]);
    }
    __syncthreads();
    if (threadIdx.x == 0) {
        *tmp_flag = atomicAdd(scratchFlag, 1);
    }
    __syncthreads();
    if (*tmp_flag == gridDim.x-1) {
        for(int i = threadIdx.x; i < C; i+= blockDim.x) {
            // todo - potentially do stochastic rounding here as well
            dbias[i] = (floatX)scratch_dbias[i];
            dweight[i] = (floatX)scratch_dweight[i];
        }
    }
}

__global__ void softmax_autoregressive_backward_kernel(floatX* dpreatt, const floatX* datt, const floatX* att,
                                                       int B, int T, int C, float scale) {
    constexpr const int BlockSize = 256;
    constexpr int T_per_block = 4;
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    __shared__ float block_acc[32];

    int idx = blockIdx.y;
    // go through blocks in reverse order, so the slowest block starts first
    int t0 = T - 1 - T_per_block*blockIdx.x;

    att += idx * T * T;
    datt += idx * T * T;
    dpreatt += idx * T * T;

    if (warp.meta_group_rank() == 0) {
        block_acc[warp.thread_rank()] = 0;
    }

    for(int to = 0; to < T_per_block; ++to) {
        int t = t0 - to;
        if(t < 0) return;
        const floatX* att_bth = att + t * T;
        const floatX* datt_bth = datt + t * T;
        floatX* dpreatt_bth = dpreatt + t * T;

        float local_sum = 0;
        for (int t2 = block.thread_rank(); t2 <= t; t2 += BlockSize) {
            local_sum += (float)att_bth[t2] * (float)datt_bth[t2];
        }

        block_acc[warp.meta_group_rank()] = cg::reduce(warp, local_sum, cg::plus<float>{});
        block.sync();
        local_sum = cg::reduce(warp, block_acc[warp.thread_rank()], cg::plus<float>{});

        for (int t3 = block.thread_rank(); t3 <= t; t3 += BlockSize) {
            // don't touch the cache. Some parts will still be here from the previous loop, and
            // we want to exploit those.
            float acc = (float)__ldcs(att_bth + t3) * ((float)__ldcs(datt_bth + t3) - local_sum);
            __stcs(dpreatt_bth + t3, (floatX)(scale * acc));
        }
    }
}

// Implements linear interpolation using only two floating-point operations (as opposed to three in a naive implementation).
// Reference: https://developer.nvidia.com/blog/lerp-faster-cuda
__device__ inline float lerp(float start, float end, float weight) {
    return fma(weight, end, fma(-weight, start, start));
}

// Termplate type T instead of floatx
template <typename Tp, typename Tg>
__global__ void adamw_kernel3(Tp* params_memory, float* master_params, Tg* grads_memory, float* m_memory, float* v_memory, size_t num_parameters,
                              float learning_rate, float beta1, float beta2, float beta1_correction, float beta2_correction, float eps, float weight_decay,
                              unsigned int seed) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   if (i >= num_parameters) return;  // guard
   float grad = (float)grads_memory[i];
   float m = m_memory[i];
   float v = v_memory[i];
   // update the first moment (momentum)
   m = lerp(grad, m, beta1);
   m_memory[i] = m;
   // update the second moment (RMSprop)
   v = lerp(grad * grad, v, beta2);
   v_memory[i] = v;
   m /= beta1_correction;  // m_hat
   v /= beta2_correction;  // v_hat
   // update the parameters (weight/bias)
   float old_param = master_params != NULL ? master_params[i] : (float)params_memory[i];
   float param = old_param - (learning_rate * (m / (sqrtf(v) + eps) + weight_decay * old_param));
   // if we have master parameters, directly update the two weight copies
    if (master_params != NULL) {
        params_memory[i] = (floatX)param; // low-precision copy, for use in the forward pass
        master_params[i] = param; // float copy, for use in the next parameter update
    } else {
        // without a master copy of params in float, do a direct update in low precision
        // and use stochastic rounding to mitigate loss of training stability
        unsigned int random = Get2dNoiseUint(threadIdx.x, blockIdx.x, seed);
        stochastic_rounding(param, &params_memory[i], random);
    }
}

struct SoftmaxParams {
    float Scale;
    float Offset;
};

__device__ SoftmaxParams prepare_softmax_blockwide_nofloat4(cg::thread_block_tile<32>& warp,
                                                   int idx, const floatX* inp, int V, int P) {
    // same but not float4
    // one row of inp, i.e. inp[idx, :] of shape (V,)

    const floatX* x = inp + idx * P;
    float thread_maxval = -INFINITY;
    float thread_sumval = 0.0f;
    // do the loop in reverse to maximise probability of L2 cache hits
    // so even small L2s get some hits on the 2nd read of the same thread
    for (int i = V + threadIdx.x - blockDim.x; i >= 0; i -= blockDim.x) {
        float v = (float)x[i];
        float old_maxval = thread_maxval;
        thread_maxval = fmaxf(thread_maxval, v);
        thread_sumval *= expf((old_maxval - thread_maxval));
        thread_sumval += expf(v - thread_maxval);
    }

    // two reductions of up to 1024 threads:
    // 1) inside warp (shuffle), 2) cross-warp (shared memory), 3) inside warp (shuffle)
    // this results in much cleaner assembly than a multi-warp cg::reduce
    __shared__ float shared_maxval[32];
    __shared__ float shared_sumval[32];
    int num_warps = blockDim.x / 32;
    int warp_id = threadIdx.x / 32;
    int lane_id = threadIdx.x % 32;

    // reduce maxval within each warp
    float warp_maxval = cg::reduce(warp, thread_maxval, cg::greater<float>{});
    // thread 0 in each warp writes to shared memory
    if (lane_id == 0) { shared_maxval[warp_id] = warp_maxval; }
    __syncthreads();
    // each thread now loads the maxval across previous warps
    // if the thread is "out of range" of data, use -FLT_MAX as the maxval
    warp_maxval = (lane_id < num_warps) ? shared_maxval[lane_id] : -FLT_MAX;
    // now reduce the maxval among the warp threads
    float block_maxval = cg::reduce(warp, warp_maxval, cg::greater<float>{});
    // each thread uses maxval to scale sumval to avoid numerical instability / overflow
    thread_sumval *= expf(thread_maxval - block_maxval);
    // (warp-level) reduce sumval, thread 0 in each warp saves result in shared memory
    float warp_sumval = cg::reduce(warp, thread_sumval, cg::plus<float>{});
    if (lane_id == 0) { shared_sumval[warp_id] = warp_sumval; }
    __syncthreads();
    // same strategy, now reduce sumval across warps
    warp_sumval = (lane_id < num_warps) ? shared_sumval[lane_id] : 0.0f;
    float block_sumval = cg::reduce(warp, warp_sumval, cg::plus<float>{});
    // return the softmax parameters
    return SoftmaxParams{1.f / block_sumval, block_maxval};
}

// same as 2 but not using float4 (see dev/cuda/classifier_fused.cu)
// will _update_ logits to logit gradients
__global__ void fused_classifier_kernel3(floatX* logits, floatX* losses, floatX* probs,
                                         const floatX* dlosses, const int* targets,
                                         int B, int T, int V, int P) {
    namespace cg = cooperative_groups;
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    int idx = blockIdx.x;
    int ix = targets[idx];

    // softmax (reading B * T * V, same logits read again below, hopefully still in cache)
    SoftmaxParams sp = prepare_softmax_blockwide_nofloat4(warp, idx, logits, V, P);

    // calculate the probability needed for the loss and update (single-threaded)
    if(threadIdx.x == 0) {
        float prob = expf((float)logits[idx * P + ix] - sp.Offset) * sp.Scale;
        losses[idx] = (floatX)(-logf(prob));
    }

    // very sensible default for dlosses is 1/(B*T), which is the uniform loss
    float dloss = (dlosses != NULL) ? (float)dlosses[idx] : 1.0f / (B*T);
    // calculate the gradients directly, saves bandwidth from probs during training
    // but also supports writing probs for inference-only and debugging
    const floatX* logits_vec = logits + idx * P;
    // note that we use the padded dimension P to access data, but we only ever
    // modify the elements up to V, ignoring the padded dimensions and leaving them at 0
    for (int i = threadIdx.x; i < V; i += blockDim.x) {
        // this is the 2nd read of logits after the one in prepare_softmax2
        // this data will never be needed again, so we reduce cache persistence
        float v = (float)__ldcs(&logits_vec[i]);
        float prob = expf(v - sp.Offset) * sp.Scale;
        if (probs != NULL) {
            probs[idx * P + i] = (floatX)prob;
        }
        float indicator = (i == ix) ? 1.0f : 0.0f;
        logits[idx * P + i] = (floatX)((prob - indicator) * dloss);
    }
}

__global__ void copy_and_cast_kernel(float* dst, const floatX* src, size_t n) {
    // a small kernel to copy and cast, i.e. `dst <- (float) src`
    const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) { dst[i] = (float)src[i]; }
}

// ----------------------------------------------------------------------------
// kernel launchers

void encoder_forward(floatX* out,
                     int* inp, floatX* wte, floatX* wpe,
                     int B, int T, int C) {
    const int N = B * T * C;
    const int block_size = 256;
    const int grid_size = CEIL_DIV(N, block_size);
    encoder_forward_kernel2<<<grid_size, block_size>>>(out, inp, wte, wpe, B, T, C);
    cudaCheck(hipGetLastError());
}

void encoder_backward(floatX* dwte, floatX* dwpe,
                    const floatX* dout, const int* inp,
                    int B, int T, int C) {
    const int N = B * T * C;
    const int block_size = 256;
    const int grid_size = CEIL_DIV(N, block_size);
    encoder_backward_kernel<<<grid_size, block_size>>>(dwte, dwpe, dout, inp, B, T, C);
    cudaCheck(hipGetLastError());
}

void layernorm_forward(floatX* out, floatX* mean, floatX* rstd,
                       floatX* inp, floatX* weight, floatX* bias,
                       int B, int T, int C) {
    const int block_size = 512;
    const int N = B * T;
    const int grid_size = CEIL_DIV(N * 32, block_size);
    layernorm_forward_kernel3<<<grid_size, block_size>>>(out, mean, rstd, inp, weight, bias, N, C);
    cudaCheck(hipGetLastError());
}

// uses cuBLASLt to fuse the bias and gelu. does not work with OC = 50257 (last layer)
// https://docs.nvidia.com/cuda/cublas/#cublasltmatmul
// https://github.com/NVIDIA/CUDALibrarySamples/blob/master/cuBLASLt/LtSgemm/sample_cublasLt_LtSgemm.cu
void matmul_forward_cublaslt(floatX* out,
                     floatX* inp, floatX* weight, floatX* bias,
                     int B, int T, int C, int OC) {
    int has_bias = (bias != NULL);

    // check bias alignment
    if(((uintptr_t)bias % 16) != 0) {
        printf("Bias pointer is not aligned (cuBLASLt requirement)!\n");
        exit(EXIT_FAILURE);
    }

    // FP16 alpha/beta need to be used if and only if HIPBLAS_COMPUTE_16F
    const float alpha = 1.0f, beta = 0.0f;
    const half alpha_fp16 = (half)alpha, beta_fp16 = (half)beta;
    const void* alpha_ptr = (CUBLAS_LOWP_COMPUTE == HIPBLAS_COMPUTE_16F) ?
                            (const void*)&alpha_fp16 : (const void*)&alpha;
    const void* beta_ptr =  (CUBLAS_LOWP_COMPUTE == HIPBLAS_COMPUTE_16F) ?
                            (const void*)&beta_fp16 : (const void*)&beta;

    int returnedResults = 0;
    hipblasLtMatmulDesc_t operationDesc;
    hipblasLtMatmulPreference_t preference;
    hipblasLtMatrixLayout_t weightLayout;
    hipblasLtMatrixLayout_t inputLayout;
    hipblasLtMatrixLayout_t outputLayout;
    hipblasLtMatrixLayout_t biasLayout;
    hipblasLtMatmulHeuristicResult_t heuristic;

    // create the operation descriptor
    hipblasOperation_t opNoTranspose = HIPBLAS_OP_N;
    hipblasOperation_t opTranspose = HIPBLAS_OP_T;
    hipblasLtEpilogue_t epilogueBias = HIPBLASLT_EPILOGUE_BIAS;

    hipDataType scale_type = (CUBLAS_LOWP_COMPUTE == HIPBLAS_COMPUTE_16F) ? HIP_R_16F : HIP_R_32F;
    cublasCheck(hipblasLtMatmulDescCreate(&operationDesc, CUBLAS_LOWP_COMPUTE, scale_type));
    cublasCheck(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &opTranspose, sizeof(opTranspose)));
    cublasCheck(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &opNoTranspose, sizeof(opNoTranspose)));
    if(has_bias) {
        cublasCheck(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogueBias,
                                                   sizeof(epilogueBias)));
    }
    cublasCheck(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bias, sizeof(bias)));

    // define matrix layouts
    cublasCheck(hipblasLtMatrixLayoutCreate(&weightLayout, CUBLAS_LOWP, C, OC, C));
    cublasCheck(hipblasLtMatrixLayoutCreate(&inputLayout, CUBLAS_LOWP, C, B*T, C));
    cublasCheck(hipblasLtMatrixLayoutCreate(&outputLayout, CUBLAS_LOWP, OC, B*T, OC));
    cublasCheck(hipblasLtMatrixLayoutCreate(&biasLayout, CUBLAS_LOWP, OC, 1, OC));

    // create a preference handle with specified max workspace
    cublasCheck(hipblasLtMatmulPreferenceCreate(&preference));
    cublasCheck(hipblasLtMatmulPreferenceSetAttribute(preference,
        HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES,
        &cublaslt_workspace_size, sizeof(cublaslt_workspace_size)));

    // find a suitable algorithm
    cublasCheck(hipblasLtMatmulAlgoGetHeuristic(cublaslt_handle, operationDesc,
        weightLayout, inputLayout, outputLayout, outputLayout,
        preference, 1, &heuristic, &returnedResults));
    if (returnedResults == 0) {
        printf("No cuBLASLt algorithm: B: %d, T: %d, C: %d, OC: %d, bias: %d\n", B, T, C, OC, has_bias);
        exit(EXIT_FAILURE);
    }

    // call the matmul
    cublasCheck(hipblasLtMatmul(cublaslt_handle, operationDesc,
        alpha_ptr, weight, weightLayout, inp, inputLayout, beta_ptr,
        out, outputLayout, out, outputLayout, &heuristic.algo,
        cublaslt_workspace, cublaslt_workspace_size, 0));

    // cleanups
    cublasCheck(hipblasLtMatmulPreferenceDestroy(preference));
    cublasCheck(hipblasLtMatmulDescDestroy(operationDesc));
    cublasCheck(hipblasLtMatrixLayoutDestroy(weightLayout));
    cublasCheck(hipblasLtMatrixLayoutDestroy(inputLayout));
    cublasCheck(hipblasLtMatrixLayoutDestroy(outputLayout));
    cublasCheck(hipblasLtMatrixLayoutDestroy(biasLayout));
}

void attention_forward(floatX* out, floatX* qkvr, floatX* att,
                       floatX* inp,
                       int B, int T, int C, int NH) {
    // Note: `inp` is not needed for backward pass, so we re-use it as a scratch buffer.
    // Its contents will be overwritten by this function.
    const int block_size = 256;
    const int softmax_block_size = 256;

    // inp is (B, T, 3C) QKV
    // preatt, att are (B, NH, T, T)
    // output is (B, T, C)
    int HS = C / NH; // head size

    // permute and separate inp from (B, T, 3, NH, HS) to 3X (B, NH, T, HS)
    floatX *q, *k, *v;
    q = qkvr + 0 * B * T * C;
    k = qkvr + 1 * B * T * C;
    v = qkvr + 2 * B * T * C;
    int total_threads = B * NH * T * HS;
    int num_blocks = CEIL_DIV(total_threads, block_size);
    permute_kernel<<<num_blocks, block_size>>>(q, k, v, inp, B, T, NH, HS);
    cudaCheck(hipGetLastError());

    // IMPORTANT: alpha/beta are FP32 for HIPBLAS_COMPUTE_32F even if FP16 inputs/outputs
    // But need FP16 scale for HIPBLAS_COMPUTE_16F (no errors otherwise, just garbage results *sigh*)
    const float alpha = 1.0f;
    const float beta = 0.0f;
    const floatX alpha_lowp = (floatX)alpha;
    const floatX beta_lowp = (floatX)beta;
    void* alpha_ptr = (CUBLAS_LOWP_COMPUTE == HIPBLAS_COMPUTE_16F) ? (void*)&alpha_lowp : (void*)&alpha;
    void* beta_ptr = (CUBLAS_LOWP_COMPUTE == HIPBLAS_COMPUTE_16F) ? (void*)&beta_lowp : (void*)&beta;

    floatX* preatt = inp;
    cublasCheck(hipblasGemmStridedBatchedEx(cublas_handle,
                                     HIPBLAS_OP_T, HIPBLAS_OP_N,
                                     T, T, HS,
                                     alpha_ptr,
                                     k, CUBLAS_LOWP, HS, T * HS,
                                     q, CUBLAS_LOWP, HS, T * HS,
                                     beta_ptr,
                                     preatt, CUBLAS_LOWP, T, T * T,
                                     B * NH,
                                     CUBLAS_LOWP_COMPUTE,
                                     HIPBLAS_GEMM_DEFAULT));

    // multiply all elements of preatt elementwise by scale
    float scale = 1.0 / sqrtf(HS);
    int grid_size = CEIL_DIV(B * NH * T * 32, softmax_block_size);
    softmax_forward_kernel5<<<grid_size, softmax_block_size>>>(att, scale, preatt, B * NH, T);
    cudaCheck(hipGetLastError());

    // new approach: first cuBLAS another batched matmul
    floatX* vaccum = inp;
    // y = att @ v # (B, nh, T, T) @ (B, nh, T, hs) -> (B, nh, T, hs)
    cublasCheck(hipblasGemmStridedBatchedEx(cublas_handle,
                                     HIPBLAS_OP_N, HIPBLAS_OP_N,
                                     HS, T, T,
                                     alpha_ptr,
                                     v, CUBLAS_LOWP, HS, T * HS,
                                     att, CUBLAS_LOWP, T, T * T,
                                     beta_ptr,
                                     vaccum, CUBLAS_LOWP, HS, T * HS,
                                     B * NH,
                                     CUBLAS_LOWP_COMPUTE,
                                     HIPBLAS_GEMM_DEFAULT));

    // now unpermute
    // y = y.transpose(1, 2).contiguous().view(B, T, C) # re-assemble all head outputs side by side
    num_blocks = CEIL_DIV(B * T * C, block_size);
    unpermute_kernel<<<num_blocks, block_size>>>(vaccum, out, B, T, NH, HS);
    cudaCheck(hipGetLastError());
}

void residual_forward(floatX* out, floatX* inp1, floatX* inp2, int N) {
    const int block_size = 256;
    const int grid_size = CEIL_DIV(N, block_size);
    residual_forward_kernel<<<grid_size, block_size>>>(out, inp1, inp2, N);
    cudaCheck(hipGetLastError());
}

void gelu_forward(floatX* out, const floatX* inp, int N) {
    const int block_size = 512;
    const int grid_size = CEIL_DIV(N, block_size * x128::size);
    gelu_forward_kernel2<<<grid_size, block_size>>>(out, inp, N);
    cudaCheck(hipGetLastError());
}

void gelu_backward(floatX* dinp, const floatX* inp, const floatX* dout, const int N) {
    const int block_size = 128;
    const int grid_size = CEIL_DIV(N, block_size);
    gelu_backward_kernel<<<grid_size, block_size>>>(dinp, inp, dout, N);
    cudaCheck(hipGetLastError());
}

void matmul_backward(floatX* dinp, floatX* dweight, floatX* dbias,
                     floatX* dout, floatX* inp, floatX* weight,
                     int B, int T, int C, int OC) {
    float one = 1.0f;
    float zero = 0.0f;
    // backward to input, uses = in the backward pass (set the gradient)
    cublasCheck(hipblasGemmEx(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, C, B*T, OC, &one,
                             weight, CUBLAS_LOWP, C, dout, CUBLAS_LOWP, OC, &zero,
                             dinp, CUBLAS_LOWP, C, CUBLAS_LOWP_COMPUTE, CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    // backward to weight, uses += in the backward pass (accumulate the gradient)
    cublasCheck(hipblasGemmEx(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, C, OC, B*T, &one,
                             inp, CUBLAS_LOWP, C, dout, CUBLAS_LOWP, OC, &one,
                             dweight, CUBLAS_LOWP, C, CUBLAS_LOWP_COMPUTE, CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    // backward to bias, if given, does a +=
    if (dbias != NULL) {
        const int block_size = 1024;
        const int grid_size = OC / 32; // for now, OC must be divisible by 32 for this kernel to work
        matmul_backward_bias_kernel4<<<grid_size, block_size, block_size * sizeof(float)>>>(dbias, dout, B, T, OC);
        cudaCheck(hipGetLastError());
    }
}

void layernorm_backward(floatX* dinp, floatX* dweight, floatX* dbias, float* scratch,
                        const floatX* dout, const floatX* inp, const floatX* weight, const floatX* mean, const floatX* rstd,
                        int B, int T, int C) {
    const int block_size = 1024;
    const int grid_size = 1 * cuda_num_SMs;
    size_t shared_mem_size = (2 * C + 1) * sizeof(float);
    hipMemset(scratch, 0, (2 * C + 1) * sizeof(float)); // todo - memset in parallel with previous kernels using streams
    layernorm_backward_kernel6<<<grid_size, block_size, shared_mem_size>>>(dinp, dweight, dbias, scratch, dout, inp, weight, mean, rstd, B, T, C);
    cudaCheck(hipGetLastError());
}

// the sequence of transformations in this compound op is:
// inp (B,T,3C) -> qkvr (B,T,3C) -> preatt (B,NH,T,T) -> att (B,NH,T,T) -> vaccum (B,T,C) -> out (B,T,C)
void attention_backward(floatX* dinp, floatX* dqkvr, floatX* dpreatt, floatX* datt, floatX* scratch,
                        const floatX* dout,
                        const floatX* qkvr, const floatX* att,
                        int B, int T, int C, int NH) {
    const int block_size = 256;
    int HS = C / NH; // head size

    // FP16 alpha/beta need to be used if and only if HIPBLAS_COMPUTE_16F
    const float alpha = 1.0f, beta = 0.0f;
    const half alpha_fp16 = (half)alpha, beta_fp16 = (half)beta;
    const void* alpha_ptr = (CUBLAS_LOWP_COMPUTE == HIPBLAS_COMPUTE_16F) ?
                            (const void*)&alpha_fp16 : (const void*)&alpha;
    const void* beta_ptr =  (CUBLAS_LOWP_COMPUTE == HIPBLAS_COMPUTE_16F) ?
                            (const void*)&beta_fp16 : (const void*)&beta;

    // unpack convenience pointers into q, k, v
    const floatX *q, *k, *v;
    q = qkvr + 0 * B * T * C;
    k = qkvr + 1 * B * T * C;
    v = qkvr + 2 * B * T * C;
    floatX *dq, *dk, *dv;
    dq = dqkvr + 0 * B * T * C;
    dk = dqkvr + 1 * B * T * C;
    dv = dqkvr + 2 * B * T * C;

    // backward through the unpermute operation
    int num_blocks = CEIL_DIV(B * T * C, block_size);
    unpermute_kernel_backward<<<num_blocks, block_size>>>(scratch, dout, B, T, NH, HS);
    cudaCheck(hipGetLastError());
    // backward into datt

    cublasCheck(hipblasGemmStridedBatchedEx(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, T, T, HS, alpha_ptr,
                                           v, CUBLAS_LOWP, HS, T * HS, scratch, CUBLAS_LOWP, HS, T * HS, beta_ptr,
                                           datt, CUBLAS_LOWP, T, T * T, B * NH, CUBLAS_LOWP_COMPUTE, HIPBLAS_GEMM_DEFAULT));

    // backward into dv
    cublasCheck(hipblasGemmStridedBatchedEx(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, HS, T, T, alpha_ptr,
                                           scratch, CUBLAS_LOWP, HS, T * HS, att, CUBLAS_LOWP, T, T * T, beta_ptr,
                                           dv, CUBLAS_LOWP, HS, T * HS, B * NH, CUBLAS_LOWP_COMPUTE, HIPBLAS_GEMM_DEFAULT));

    // backward into preatt
    int hs = C / NH; // head size
    float scale = 1.0f / sqrtf(hs);
    softmax_autoregressive_backward_kernel<<<dim3(T / 4, B * NH), 256>>>(dpreatt, datt, att, B, T, C, scale);
    cudaCheck(hipGetLastError());
    // backward into q
    cublasCheck(hipblasGemmStridedBatchedEx(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, HS, T, T, alpha_ptr,
                                           k, CUBLAS_LOWP, HS, T * HS, dpreatt, CUBLAS_LOWP, T, T * T, beta_ptr,
                                           dq, CUBLAS_LOWP, HS, T * HS, B * NH, CUBLAS_LOWP_COMPUTE, HIPBLAS_GEMM_DEFAULT));
    // backward into k
    cublasCheck(hipblasGemmStridedBatchedEx(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, HS, T, T, alpha_ptr,
                                           q, CUBLAS_LOWP, HS, T * HS, dpreatt, CUBLAS_LOWP, T, T * T, beta_ptr,
                                           dk, CUBLAS_LOWP, HS, T * HS, B * NH, CUBLAS_LOWP_COMPUTE, HIPBLAS_GEMM_DEFAULT));
    // backward into inp
    num_blocks = CEIL_DIV(B * NH * T * HS, block_size);
    permute_kernel_backward<<<num_blocks, block_size>>>(dinp, dq, dk, dv, B, T, NH, HS);
    cudaCheck(hipGetLastError());
}

// replaces logits with logit gradients
template <typename Type>
void fused_classifier3(Type* logits, Type* losses,
                      const Type* dlosses, const int* targets,
                      int B, int T, int V, int P) {
    const int block_size = 1024;
    const int N = B * T;
    const int grid_size = N;
    fused_classifier_kernel3<<<grid_size, block_size>>>(logits, losses, (Type*)NULL, dlosses, targets, B, T, V, P);
    cudaCheck(hipGetLastError());
}

// ----------------------------------------------------------------------------
// GPT-2 model definition

typedef struct {
    int max_seq_len; // max sequence length, e.g. 1024
    int vocab_size; // vocab size, e.g. 50257
    int padded_vocab_size; // padded to e.g. %128==0, 50304
    int num_layers; // number of layers, e.g. 12
    int num_heads; // number of heads in attention, e.g. 12
    int channels; // number of channels, e.g. 768
} GPT2Config;

// the parameters of the model
constexpr const int NUM_PARAMETER_TENSORS = 16;
typedef struct {
    floatX* wte; // (V, C)
    floatX* wpe; // (maxT, C)
    floatX* ln1w; // (L, C)
    floatX* ln1b; // (L, C)
    floatX* qkvw; // (L, 3*C, C)
    floatX* qkvb; // (L, 3*C)
    floatX* attprojw; // (L, C, C)
    floatX* attprojb; // (L, C)
    floatX* ln2w; // (L, C)
    floatX* ln2b; // (L, C)
    floatX* fcw; // (L, 4*C, C)
    floatX* fcb; // (L, 4*C)
    floatX* fcprojw; // (L, C, 4*C)
    floatX* fcprojb; // (L, C)
    floatX* lnfw; // (C)
    floatX* lnfb; // (C)
} ParameterTensors;
static_assert(sizeof(ParameterTensors) == NUM_PARAMETER_TENSORS * sizeof(void*), "Inconsistent sizes!");

void fill_in_parameter_sizes(size_t* param_sizes, size_t* param_sizeof, GPT2Config config) {
    size_t Vp = config.padded_vocab_size;
    size_t C = config.channels;
    size_t maxT = config.max_seq_len;
    size_t L = config.num_layers;
    param_sizes[0] = Vp * C; // wte
    param_sizes[1] = maxT * C; // wpe
    param_sizes[2] = L * C; // ln1w
    param_sizes[3] = L * C; // ln1b
    param_sizes[4] = L * (3 * C) * C; // qkvw
    param_sizes[5] = L * (3 * C); // qkvb
    param_sizes[6] = L * C * C; // attprojw
    param_sizes[7] = L * C; // attprojb
    param_sizes[8] = L * C; // ln2w
    param_sizes[9] = L * C; // ln2b
    param_sizes[10] = L * (4 * C) * C; // fcw
    param_sizes[11] = L * (4 * C); // fcb
    param_sizes[12] = L * C * (4 * C); // fcprojw
    param_sizes[13] = L * C; // fcprojb
    param_sizes[14] = C; // lnfw
    param_sizes[15] = C; // lnfb

    // populate the parameter sizes in bytes (all the same for now, keeping for future use)
    for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        param_sizeof[i] = sizeof(floatX);
    }
}

// allocate memory for the parameters and point the individual tensors to the right places
void* malloc_and_point_parameters(ParameterTensors* params, size_t* param_elements, size_t *param_sizeof) {
    // calculate the total number of parameters and bytes across all tensors
    size_t num_parameters = 0;
    size_t num_parameters_bytes = 0;
    for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        num_parameters += param_elements[i];
        num_parameters_bytes += param_elements[i] * param_sizeof[i];
    }
    // malloc all parameters all at once on the device
    void* params_memory;
    cudaCheck(hipMalloc((void**)&params_memory, num_parameters_bytes));
    // assign all the tensors their place in the array
    floatX** ptrs[] = {
        &params->wte, &params->wpe, (floatX**)&params->ln1w, (floatX**)&params->ln1b, &params->qkvw, &params->qkvb,
        &params->attprojw, &params->attprojb, (floatX**)&params->ln2w, (floatX**)&params->ln2b, &params->fcw, &params->fcb,
        &params->fcprojw, &params->fcprojb, (floatX**)&params->lnfw, (floatX**)&params->lnfb
    };
    char* params_memory_iterator = (char*)params_memory;
    for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        *(ptrs[i]) = (floatX*)params_memory_iterator;
        params_memory_iterator += param_elements[i] * param_sizeof[i];
    }
    return params_memory;
}

#define NUM_ACTIVATION_TENSORS 21
typedef struct {
    floatX* encoded; // (B, T, C)
    floatX* ln1; // (L, B, T, C)
    floatX* ln1_mean; // (L, B, T)
    floatX* ln1_rstd; // (L, B, T)
    floatX* atty; // (L, B, T, C)
    floatX* att; // (L, B, NH, T, T) (smaller with cuDNN)
    floatX* attproj; // (L, B, T, C)
    floatX* residual2; // (L, B, T, C)
    floatX* ln2; // (L, B, T, C)
    floatX* ln2_mean; // (L, B, T)
    floatX* ln2_rstd; // (L, B, T)
    floatX* fch; // (L, B, T, 4*C)
    floatX* fch_gelu; // (L, B, T, 4*C)
    floatX* fcproj; // (L, B, T, C)
    floatX* residual3; // (L, B, T, C)
    floatX* lnf; // (B, T, C)
    floatX* lnf_mean; // (B, T)
    floatX* lnf_rstd; // (B, T)
    floatX* losses; // (B, T)
    // adding these two compared to the CPU .c code, needed for attention kernel as buffers
    floatX* qkvr; // (L, B, T, 3*C)
    // in inference mode, this buffer will store the logits
    // in training mode, this buffer will contain the *gradients* of the logits.
    // during the processing of transformer blocks, we will also use this as a
    // general scratchpad buffer. Allocation is made large enough to hold (B, T, 3C),
    // (B, NH, T, T), and (B, T, V) shaped tensors.
    floatX* output;
} ActivationTensors;

void fill_in_activation_sizes(size_t* act_sizes, size_t B, size_t T, GPT2Config config) {
    size_t Vp = config.padded_vocab_size;
    size_t L = config.num_layers;
    size_t NH = config.num_heads;
    size_t C = config.channels;
    act_sizes[0] = B * T * C; // encoded
    act_sizes[1] = L * B * T * C; // ln1
    act_sizes[2] = L * B * T; // ln1_mean
    act_sizes[3] = L * B * T; // ln1_rstd
    act_sizes[4] = L * B * T * C; // atty
    #ifdef ENABLE_CUDNN
    // FP32 stats tensor for cuDNN to be passed to backward pass
    act_sizes[5] = L * B * NH * T * (sizeof(float) / sizeof(floatX));
    #else
    act_sizes[5] = L * B * NH * T * T; // att
    #endif
    act_sizes[6] = L * B * T * C; // attproj
    act_sizes[7] = L * B * T * C; // residual2
    act_sizes[8] = L * B * T * C; // ln2
    act_sizes[9] = L * B * T; // ln2_mean
    act_sizes[10] = L * B * T; // ln2_rstd
    act_sizes[11] = L * B * T * 4*C; // fch
    act_sizes[12] = L * B * T * 4*C; // fch_gelu
    act_sizes[13] = L * B * T * C; // fcproj
    act_sizes[14] = L * B * T * C; // residual3
    act_sizes[15] = B * T * C; // lnf
    act_sizes[16] = B * T; // lnf_mean
    act_sizes[17] = B * T; // lnf_rstd
    act_sizes[18] = B * T; // losses
    act_sizes[19] = L * B * T * 3*C; // qkvr
    act_sizes[20] = B * T * max(3*C, max(NH*T, Vp)); // output / scratch
}

// Backward pass is conceptually quite different from forward, because we can discard
// the activations of a layer as soon as we're done with it. This lets us aggressively
// reuse memory, so that we need far fewer tensors for backward state.
#ifdef ENABLE_CUDNN
#define NUM_BACKWARD_TENSORS 2
#else
#define NUM_BACKWARD_TENSORS 3
#endif

typedef struct {
    floatX* bt4c; // (B, T, 4*C)
    floatX* residual3; // (B, T, C)
    #ifndef ENABLE_CUDNN
    floatX* preatt; // (B, NH, T, T)
    #endif
} GradActTensors;

void fill_in_grad_act_sizes(size_t* act_sizes, size_t B, size_t T, GPT2Config config) {
    size_t C = config.channels;
    act_sizes[0] = B * T * 4 * C; // bt4c
    act_sizes[1] = B * T * C; // residual3

    #ifndef ENABLE_CUDNN
    size_t NH = config.num_heads;
    act_sizes[2] = B * NH * T * T; // preatt
    #endif
}

void* malloc_and_point(floatX** targets[], const size_t* act_sizes, size_t n) {
    size_t num_activations = 0;
    for (size_t i = 0; i < n; i++) {
        num_activations += act_sizes[i];
    }
    void* acts_memory;
    cudaCheck(hipMalloc((void**)&acts_memory, num_activations * sizeof(floatX)));
    char* acts_memory_iterator = (char*)acts_memory;
    for (size_t i = 0; i < n; i++) {
        *(targets[i]) = (floatX*)acts_memory_iterator;
        acts_memory_iterator += act_sizes[i] * sizeof(floatX);
    }
    return acts_memory;
}

void* malloc_and_point_activations(ActivationTensors* acts, const size_t* act_sizes) {
    floatX** ptrs[] = {
        &acts->encoded, &acts->ln1, &acts->ln1_mean, &acts->ln1_rstd, &acts->atty,
        &acts->att, &acts->attproj, &acts->residual2, &acts->ln2, &acts->ln2_mean,
        &acts->ln2_rstd, &acts->fch, &acts->fch_gelu, &acts->fcproj, &acts->residual3, &acts->lnf,
        &acts->lnf_mean, &acts->lnf_rstd, &acts->losses, &acts->qkvr, &acts->output
    };
    return malloc_and_point(ptrs, act_sizes, NUM_ACTIVATION_TENSORS);
}

void* malloc_and_point_backward(GradActTensors* acts, const size_t* act_sizes) {
    floatX** ptrs[] = {
        &acts->bt4c, &acts->residual3,
        #ifndef ENABLE_CUDNN
        &acts->preatt,
        #endif
    };
    return malloc_and_point(ptrs, act_sizes, NUM_BACKWARD_TENSORS);
}

typedef struct {
    GPT2Config config;
    // the weights of the model, and their sizes
    ParameterTensors params;
    size_t param_elements[NUM_PARAMETER_TENSORS];
    size_t param_sizeof[NUM_PARAMETER_TENSORS];
    void* params_memory;
    size_t num_parameters;
    size_t num_parameters_bytes;
    // gradients of the weights
    ParameterTensors grads;
    void* grads_memory;
    // buffers for the AdamW optimizer
    float* m_memory;
    float* v_memory;
    float* master_weights;     // is NULL unless fp32 weights is enabled.
    // the activations of the model, and their sizes
    ActivationTensors acts;
    size_t act_sizes[NUM_ACTIVATION_TENSORS];
    void* acts_memory;
    size_t num_activations;
    // gradients of the activations
    GradActTensors grads_acts;
    size_t num_grad_acts;
    void* grads_acts_memory;
    // other run state configuration
    int batch_size; // the batch size (B) of current forward pass
    int seq_len; // the sequence length (T) of current forward pass
    int* inputs; // the input tokens for the current forward pass
    int* targets; // the target tokens for the current forward pass
    float mean_loss; // after a forward pass with targets, will be populated with the mean loss
    float accumulated_mean_loss; // Mean loss after aggregating it on all GPUs
    floatX* cpu_losses; // CPU buffer to copy the losses to, allocated with hipHostMalloc
    unsigned long long rng_state; // the RNG state for seeding stochastic rounding etc.
    int use_master_weights;
} GPT2;

void gpt2_build_from_checkpoint(GPT2 *model, const char* checkpoint_path) {

    if (PRECISION_MODE == PRECISION_FP16) {
        // TODO for later perhaps, would require us dynamically converting the
        // model weights from fp32 to fp16 online, here in this function, or writing
        // the fp16 weights directly from Python, which we only do for fp32/bf16 atm.
        fprintf(stderr, "build_from_checkpoint() does not support fp16 right now.\n");
        exit(EXIT_FAILURE);
    }

    // read in model from a checkpoint file
    FILE *model_file = fopenCheck(checkpoint_path, "rb");
    int model_header[256];
    freadCheck(model_header, sizeof(int), 256, model_file);
    if (model_header[0] != 20240326) { printf("Bad magic model file\n"); exit(EXIT_FAILURE); }
    int version = model_header[1];
    if (!(version == 3 || version == 5)) {
        // 3 = fp32, padded vocab
        // 5 = bf16, padded vocab, layernorms also in bf16
        fprintf(stderr, "Bad version in model file\n");
        fprintf(stderr, "---> HINT: try to re-run `python train_gpt2.py`\n");
        exit(EXIT_FAILURE);
    }

    // read in hyperparameters
    model->config.max_seq_len = model_header[2];
    model->config.vocab_size = model_header[3];
    model->config.num_layers = model_header[4];
    model->config.num_heads = model_header[5];
    model->config.channels = model_header[6];
    model->config.padded_vocab_size = model_header[7];

    // allocate space for all the parameters and read them in
    fill_in_parameter_sizes(model->param_elements, model->param_sizeof, model->config);

    model->num_parameters = 0;
    model->num_parameters_bytes = 0;
    for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        model->num_parameters += model->param_elements[i];
        model->num_parameters_bytes += model->param_elements[i] * model->param_sizeof[i];
    }

    // create memory for model parameters on the device
    model->params_memory = malloc_and_point_parameters(&model->params, model->param_elements, model->param_sizeof);

    // read in all the parameters from file and copy them to device
    float* params_memory_cpu = (float*)mallocCheck(model->num_parameters_bytes);
    freadCheck(params_memory_cpu, 1, model->num_parameters_bytes, model_file);
    cudaCheck(hipMemcpy(model->params_memory, params_memory_cpu, model->num_parameters_bytes, hipMemcpyHostToDevice));
    free(params_memory_cpu);
    fcloseCheck(model_file);

    // other inits
    model->acts_memory = NULL;
    model->grads_memory = NULL;
    model->m_memory = NULL;
    model->v_memory = NULL;
    model->master_weights = NULL;
    model->grads_acts_memory = NULL;
    model->inputs = NULL;
    model->targets = NULL;
    model->cpu_losses = NULL;
    model->batch_size = 0;
    model->seq_len = 0;
    model->mean_loss = -1.0f; // -1.0f will designate no loss
    model->rng_state = 13371337;
    model->use_master_weights = 1; // keep master weights copy in float for optim update?
}

void gpt2_forward(GPT2 *model, int* inputs, int* targets, size_t B, size_t T) {
    // targets are optional and could be NULL
    // in this function we must be careful and use size_t instead of int, otherwise
    // we could overflow int. E.g. l * B * NH * T * T overflows int at B 16.

    // ensure the model was initialized or error out
    if (model->params_memory == NULL) {
        printf("Error: model was not initialized properly.\n");
        exit(EXIT_FAILURE);
    }

    // convenience parameters
    size_t V = model->config.vocab_size;
    size_t Vp = model->config.padded_vocab_size;
    size_t L = model->config.num_layers;
    size_t NH = model->config.num_heads;
    size_t C = model->config.channels;

    // validate inputs, all indices must be in the range [0, V)
    for(int i = 0; i < B * T; i++) {
        assert(0 <= inputs[i] && inputs[i] < V);
        if (targets != NULL) {
            assert(0 <= targets[i] && targets[i] < V);
        }
    }

    // allocate space for all the activations if needed (done here, lazily)
    if(model->acts_memory == NULL) {
        // record the current B,T as well
        model->batch_size = B;
        model->seq_len = T;
        // allocate the space
        fill_in_activation_sizes(model->act_sizes, B, T, model->config);
        size_t num_activations = 0;
        for (size_t i = 0; i < NUM_ACTIVATION_TENSORS; i++) {
            num_activations += model->act_sizes[i];
        }
        model->num_activations = num_activations;
        model->acts_memory = malloc_and_point_activations(&model->acts, model->act_sizes);
        printf0("allocated %d MiB for activations\n", (int)round(num_activations * sizeof(floatX) / (1024 * 1024)));
        // also create memory for caching inputs and targets
        cudaCheck(hipMalloc((void**)&model->inputs, B * T * sizeof(int)));
        cudaCheck(hipMalloc((void**)&model->targets, B * T * sizeof(int)));
        cudaCheck(hipHostMalloc((void**)&model->cpu_losses, B * T * sizeof(floatX)));
    } else {
        // validate B,T is consistent with how we've allocated the memory before
        // in principle we could get more clever here in the future, for now this is safest
        if (B != model->batch_size || T != model->seq_len) {
            printf("Model: B=%d T=%d, Desired: B=%d T=%d\n", model->batch_size, model->seq_len, (int)B, (int)T);
            exit(EXIT_FAILURE);
        }
    }

    // copy inputs/targets to the model
    cudaCheck(hipMemcpy(model->inputs, inputs, B * T * sizeof(int), hipMemcpyHostToDevice));
    if (targets != NULL) {
        cudaCheck(hipMemcpy(model->targets, targets, B * T * sizeof(int), hipMemcpyHostToDevice));
    }

    // forward pass
    ParameterTensors params = model->params; // for brevity
    ActivationTensors acts = model->acts;
    floatX* residual;
    encoder_forward(acts.encoded, model->inputs, params.wte, params.wpe, B, T, C); // encoding goes into residual[0]

    for (int l = 0; l < L; l++) {

        residual = l == 0 ? acts.encoded : acts.residual3 + (l-1) * B * T * C;

        // get the pointers of the weights for this layer
        floatX* l_ln1w = params.ln1w + l * C;
        floatX* l_ln1b = params.ln1b + l * C;
        floatX* l_qkvw = params.qkvw + l * 3*C * C;
        floatX* l_qkvb = params.qkvb + l * 3*C;
        floatX* l_attprojw = params.attprojw + l * C * C;
        floatX* l_attprojb = params.attprojb + l * C;
        floatX* l_ln2w = params.ln2w + l * C;
        floatX* l_ln2b = params.ln2b + l * C;
        floatX* l_fcw = params.fcw + l * 4*C * C;
        floatX* l_fcb = params.fcb + l * 4*C;
        floatX* l_fcprojw = params.fcprojw + l * C * 4*C;
        floatX* l_fcprojb = params.fcprojb + l * C;

        // get the pointers of the activations for this layer
        floatX* l_ln1 = acts.ln1 + l * B * T * C;
        floatX* l_ln1_mean = acts.ln1_mean + l * B * T;
        floatX* l_ln1_rstd = acts.ln1_rstd + l * B * T;
        floatX* l_qkvr = acts.qkvr + l * B * T * 3*C;
        floatX* l_atty = acts.atty + l * B * T * C;
        floatX* l_attproj = acts.attproj + l * B * T * C;
        floatX* l_residual2 = acts.residual2 + l * B * T * C;
        floatX* l_ln2 = acts.ln2 + l * B * T * C;
        floatX* l_ln2_mean = acts.ln2_mean + l * B * T;
        floatX* l_ln2_rstd = acts.ln2_rstd + l * B * T;
        floatX* l_fch = acts.fch + l * B * T * 4*C;
        floatX* l_fch_gelu = acts.fch_gelu + l * B * T * 4*C;
        floatX* l_fcproj = acts.fcproj + l * B * T * C;
        floatX* l_residual3 = acts.residual3 + l * B * T * C;

        // now do the forward pass
        layernorm_forward(l_ln1, l_ln1_mean, l_ln1_rstd, residual, l_ln1w, l_ln1b, B, T, C);

        #ifdef ENABLE_CUDNN
        float* l_att = (float*)acts.att + l * B * NH * T; // cuDNN needs a smaller FP32 tensor
        matmul_forward_cublaslt(l_qkvr, l_ln1, l_qkvw, l_qkvb, B, T, C, 3*C);
        attention_forward_cudnn(l_atty, (float*)l_att, l_qkvr, B, T, NH, C);
        #else
        floatX* l_att = acts.att + l * B * NH * T * T;
        // these are only needed as scratchpads for the forward pass, but
        // need not be stored for backward
        floatX* scratch = (floatX*)acts.output;
        matmul_forward_cublaslt(scratch, l_ln1, l_qkvw, l_qkvb, B, T, C, 3*C);
        attention_forward(l_atty, l_qkvr, l_att, scratch, B, T, C, NH);
        #endif

        matmul_forward_cublaslt(l_attproj, l_atty, l_attprojw, l_attprojb, B, T, C, C);
        residual_forward(l_residual2, residual, l_attproj, B*T*C);
        layernorm_forward(l_ln2, l_ln2_mean, l_ln2_rstd, l_residual2, l_ln2w, l_ln2b, B, T, C);
        matmul_forward_cublaslt(l_fch, l_ln2, l_fcw, l_fcb, B, T, C, 4*C);
        gelu_forward(l_fch_gelu, l_fch, B*T*4*C);
        matmul_forward_cublaslt(l_fcproj, l_fch_gelu, l_fcprojw, l_fcprojb, B, T, 4*C, C);
        residual_forward(l_residual3, l_residual2, l_fcproj, B*T*C);
    }

    residual = acts.residual3 + (L-1) * B * T * C; // last residual is in residual3
    layernorm_forward(acts.lnf, acts.lnf_mean, acts.lnf_rstd, residual, params.lnfw, params.lnfb, B, T, C);
    matmul_forward_cublaslt(acts.output, acts.lnf, params.wte, NULL, B, T, C, Vp);

    // also forward the cross-entropy loss function if we have the targets
    if (targets != NULL) {
        // fused classifier: does the forward pass and first part of the backward pass
        // we're passing dlosses = NULL, which will default them to 1.0f/(B*T), i.e. uniform loss
        fused_classifier3(acts.output, acts.losses, (floatX*)NULL, model->targets, B, T, V, Vp);
        // for convenience also evaluate the mean loss (TODO re-think this compute+sync point)
        // move the (B,T) losses to CPU
        cudaCheck(hipMemcpy(model->cpu_losses, acts.losses, B * T * sizeof(floatX), hipMemcpyDeviceToHost));
        float mean_loss = 0.0f;
        for (int i=0; i<B*T; i++) { mean_loss += (float)(model->cpu_losses[i]); }
        mean_loss /= B*T;
        model->mean_loss = mean_loss;

    } else {
        // if we don't have targets, we don't have loss
        model->mean_loss = -1.0f;
    }
}

void gpt2_zero_grad(GPT2 *model) {
    if (model->grads_acts_memory != NULL) { cudaCheck(hipMemset(model->grads_acts_memory, 0, model->num_grad_acts * sizeof(floatX))); }
    if (model->grads_memory != NULL) { cudaCheck(hipMemset(model->grads_memory, 0, model->num_parameters * sizeof(floatX))); }
}

void gpt2_backward(GPT2 *model) {
    // double check we forwarded previously, with targets
    if (model->mean_loss == -1.0f) {
        printf("Error: must forward with targets before backward\n");
        exit(EXIT_FAILURE);
    }

    // lazily allocate the memory for gradients of the weights and activations, if needed
    if (model->grads_memory == NULL) {
        // allocate buffers for weight gradients
        model->grads_memory = malloc_and_point_parameters(&model->grads, model->param_elements, model->param_sizeof);
        printf0("allocated %d MiB for parameter gradients\n", (int)round(model->num_parameters * sizeof(floatX) / (1024 * 1024)));
        // we're going to be clever for the activations backward pass. we don't need to exactly
        // mirror the forward pass activations and we will save memory.
        size_t bw_act_sizes[NUM_ACTIVATION_TENSORS];
        fill_in_grad_act_sizes(bw_act_sizes, model->batch_size, model->seq_len, model->config);
        // count up and allocate the space
        model->grads_acts_memory = malloc_and_point_backward(&model->grads_acts, bw_act_sizes);
        model->num_grad_acts = 0;
        for (size_t i = 0; i < NUM_BACKWARD_TENSORS; i++) {
            model->num_grad_acts += bw_act_sizes[i];
        }
        printf0("allocated %d MiB for activation gradients\n", (int)round(model->num_grad_acts * sizeof(floatX) / (1024 * 1024)));
        // init gradients of parameters and activations to zero
        gpt2_zero_grad(model);
    }

    // convenience shortcuts, size_t instead of int so that pointer arithmetics don't overflow
    size_t B = model->batch_size;
    size_t T = model->seq_len;
    size_t Vp = model->config.padded_vocab_size;
    size_t L = model->config.num_layers;
    size_t NH = model->config.num_heads;
    size_t C = model->config.channels;

    // backward pass: go in the reverse order of the forward pass, and call backward() functions
    ParameterTensors params = model->params; // for brevity
    ParameterTensors grads = model->grads;
    ActivationTensors acts = model->acts;
    GradActTensors grads_acts = model->grads_acts;

    // re-use the output buffer of the forward pass as a scratchpad during backward pass
    float*  scratchF = (float*)acts.output;

    // we kick off the chain rule by filling in dlosses with 1.0f/(B*T)
    // this was done in the fused classifier kernel as last step of forward pass
    // technically that is a small, inline backward() pass of calculating
    // total, final loss as the mean over all losses over all (B,T) positions in the batch
    // next: backward the classifier matmul
    matmul_backward(grads_acts.bt4c, grads.wte, NULL, acts.output, acts.lnf, params.wte, B, T, C, Vp);
    // backward the final layernorm
    floatX* residual = acts.residual3 + (L-1) * B * T * C; // last residual is in residual3
    floatX* dresidual = (floatX*)grads_acts.residual3; // the main buffer holding the gradient in the backward pass
    layernorm_backward(dresidual, grads.lnfw, grads.lnfb, scratchF, grads_acts.bt4c, residual, params.lnfw, acts.lnf_mean, acts.lnf_rstd, B, T, C);

    // now backward all the layers
    for (int l = L-1; l >= 0; l--) {
        residual = l == 0 ? acts.encoded : acts.residual3 + (l-1) * B * T * C;

        // get the pointers of the weights for this layer
        floatX* l_ln1w = params.ln1w + l * C;
        floatX* l_qkvw = params.qkvw + l * 3*C * C;
        floatX* l_attprojw = params.attprojw + l * C * C;
        floatX* l_ln2w = params.ln2w + l * C;
        floatX* l_fcw = params.fcw + l * 4*C * C;
        floatX* l_fcprojw = params.fcprojw + l * C * 4*C;
        // get the pointers of the gradients of the weights for this layer
        floatX* dl_ln1w = grads.ln1w + l * C;
        floatX* dl_ln1b = grads.ln1b + l * C;
        floatX* dl_qkvw = grads.qkvw + l * 3*C * C;
        floatX* dl_qkvb = grads.qkvb + l * 3*C;
        floatX* dl_attprojw = grads.attprojw + l * C * C;
        floatX* dl_attprojb = grads.attprojb + l * C;
        floatX* dl_ln2w = grads.ln2w + l * C;
        floatX* dl_ln2b = grads.ln2b + l * C;
        floatX* dl_fcw = grads.fcw + l * 4*C * C;
        floatX* dl_fcb = grads.fcb + l * 4*C;
        floatX* dl_fcprojw = grads.fcprojw + l * C * 4*C;
        floatX* dl_fcprojb = grads.fcprojb + l * C;
        // get the pointers of the activations for this layer
        floatX* l_ln1 = acts.ln1 + l * B * T * C;
        floatX* l_ln1_mean = acts.ln1_mean + l * B * T;
        floatX* l_ln1_rstd = acts.ln1_rstd + l * B * T;
        floatX* l_qkvr = acts.qkvr + l * B * T * 3*C;
        floatX* l_atty = acts.atty + l * B * T * C;
        floatX* l_residual2 = acts.residual2 + l * B * T * C;
        floatX* l_ln2 = acts.ln2 + l * B * T * C;
        floatX* l_ln2_mean = acts.ln2_mean + l * B * T;
        floatX* l_ln2_rstd = acts.ln2_rstd + l * B * T;
        floatX* l_fch = acts.fch + l * B * T * 4*C;
        floatX* l_fch_gelu = acts.fch_gelu + l * B * T * 4*C;
        // get the pointers of the gradients of the activations for this layer
        // notice that there is no l *, because we just have a single copy, and keep
        // re-using this memory in every Transformer block as we calculate backward pass

        // we need a B x T x C buffer; thankfully, the forward activation for lnf isn't needed anymore,
        // so we can co-opt it here.
        floatX* dl_btc = (floatX*)acts.lnf;
        floatX* dl_bt4c = (floatX*)grads_acts.bt4c;

        // backprop this layer
        matmul_backward(dl_bt4c, dl_fcprojw, dl_fcprojb, dresidual, l_fch_gelu, l_fcprojw, B, T, 4*C, C);
        gelu_backward(dl_bt4c, l_fch, dl_bt4c, B*T*4*C);
        matmul_backward(dl_btc, dl_fcw, dl_fcb, dl_bt4c, l_ln2, l_fcw, B, T, C, 4 * C);
        // layernorm backward does += to the dresidual, so it correctly accumulates grad from the MLP block above
        layernorm_backward(dresidual, dl_ln2w, dl_ln2b, scratchF, dl_btc, l_residual2, l_ln2w, l_ln2_mean, l_ln2_rstd, B, T, C);
        matmul_backward(dl_btc, dl_attprojw, dl_attprojb, dresidual, l_atty, l_attprojw, B, T, C, C);

        #ifdef ENABLE_CUDNN
        float* l_att = (float*)acts.att + l * B * NH * T; // cuDNN needs a smaller FP32 tensor
        attention_backward_cudnn(dl_bt4c, dl_btc, l_qkvr, l_atty, (float*)l_att, B, T, NH, C);
        #else
        floatX* l_att = acts.att + l * B * NH * T * T;
        // we need B x T x (4)C buffers. l_atty and l_fch aren't needed anymore at this point, so reuse their memory
        floatX* buffer_a = l_atty;
        floatX* buffer_b = l_fch;        // this is B x T x 4C, so even larger than what we need
        floatX* dl_preatt = (floatX*)grads_acts.preatt; // dedicated scratchpad allocation
        floatX* scratchX =  (floatX*)acts.output;
        attention_backward(dl_bt4c, buffer_b, dl_preatt, scratchX, buffer_a, dl_btc, l_qkvr, l_att, B, T, C, NH);
        #endif

        // QKV parameter gradients
        matmul_backward(dl_btc, dl_qkvw, dl_qkvb, dl_bt4c, l_ln1, l_qkvw, B, T, C, 3 * C);
        // layernorm backward does += to dresidual, so it correctly accumulates gradient for the Attention block above
        layernorm_backward(dresidual, dl_ln1w, dl_ln1b, scratchF, dl_btc, residual, l_ln1w, l_ln1_mean, l_ln1_rstd, B, T, C);
    }
    encoder_backward(grads.wte, grads.wpe, dresidual, model->inputs, B, T, C);
}

// Compute a mean of a single CPU value across all GPU processes. No-op when multi-GPU is disabled.
float multi_gpu_cpu_float_mean(float value, const MultiGpuConfig* multi_gpu_config) {
#ifdef MULTI_GPU
    // MPI doesn't support all reduce with mean, so we sum up, then divide.
    float result;
    mpiCheck(MPI_Allreduce(&value, &result, 1, MPI_FLOAT, MPI_SUM, MPI_COMM_WORLD));
    return result / multi_gpu_config->num_processes;
#else
    return value;
#endif
}

// Averages out the loss and gradients across all GPUs. No-op when multi-GPU is disabled.
// todo - this version only works if all the parameters are the same size (floatX)
void gpt2_multi_gpu_accumulate(GPT2* model, MultiGpuConfig* multi_gpu_config) {
    // Average all losses.
    model->accumulated_mean_loss = multi_gpu_cpu_float_mean(model->mean_loss, multi_gpu_config);
#ifdef MULTI_GPU
    // Average all gradients.
    ncclCheck(ncclAllReduce(model->grads_memory, model->grads_memory,
        model->num_parameters,
        ncclFloatX, ncclAvg,
        multi_gpu_config->nccl_comm,
        // use 0 for default stream (all other computations use this stream)
        /*stream=*/0));
#endif
}

void gpt2_update(GPT2 *model, float learning_rate, float beta1, float beta2, float eps, float weight_decay, int t) {
    // reference: https://pytorch.org/docs/stable/generated/torch.optim.AdamW.html

    // lazily allocate the memory for m_memory and v_memory
    if (model->m_memory == NULL) {
        cudaCheck(hipMalloc((void**)&model->m_memory, model->num_parameters * sizeof(float)));
        cudaCheck(hipMalloc((void**)&model->v_memory, model->num_parameters * sizeof(float)));
        cudaCheck(hipMemset(model->m_memory, 0, model->num_parameters * sizeof(float)));
        cudaCheck(hipMemset(model->v_memory, 0, model->num_parameters * sizeof(float)));
        printf0("allocated %zu MiB for AdamW optimizer state m\n", (model->num_parameters * sizeof(float)) >> 20);
        printf0("allocated %zu MiB for AdamW optimizer state v\n", (model->num_parameters * sizeof(float)) >> 20);
        if (model->use_master_weights == 1) {
            // allocate one more buffer to keep the master copy of weights as float, and copy the weights over
            cudaCheck(hipMalloc((void**)&model->master_weights, model->num_parameters * sizeof(float)));
            copy_and_cast_kernel<<<CEIL_DIV(model->num_parameters, 512), 512>>>(model->master_weights, (floatX*)model->params_memory, model->num_parameters);
            cudaCheck(hipGetLastError());
            printf0("allocated %zu MiB for master copy of params\n", (model->num_parameters * sizeof(float)) >> 20);
        }
    }

    int block_size = 512;
    int num_blocks = CEIL_DIV(model->num_parameters, block_size);
    float beta1_correction = 1.0f - powf(beta1, t);
    float beta2_correction = 1.0f - powf(beta2, t);
    unsigned int seed = random_u32(&model->rng_state);
    adamw_kernel3<<<num_blocks, block_size>>>((floatX*)model->params_memory, model->master_weights,
                                              (floatX*)model->grads_memory, model->m_memory, model->v_memory,
                                              model->num_parameters,
                                              learning_rate, beta1, beta2, beta1_correction, beta2_correction, eps, weight_decay, seed);
    cudaCheck(hipGetLastError());
}

void gpt2_free(GPT2 *model) {
    cudaCheck(hipFree(model->params_memory));
    cudaCheck(hipFree(model->grads_memory));
    cudaCheck(hipFree(model->m_memory));
    cudaCheck(hipFree(model->v_memory));
    cudaCheck(hipFree(model->master_weights));
    cudaCheck(hipFree(model->acts_memory));
    cudaCheck(hipFree(model->grads_acts_memory));
    cudaCheck(hipFree(model->inputs));
    cudaCheck(hipFree(model->targets));
    hipHostFree(model->cpu_losses);
}

#ifndef TESTING
// if we are TESTING (see test_gpt2.cu), we'll skip the int main below

// ----------------------------------------------------------------------------
// data loader lite: returns random batches of data from a file of integers

typedef struct {
    // Distributed data parallel specifics.
    // Each worker loads it's own chunk of data.
    int process_rank;
    int num_processes;
    // hyperparameters. use size_t to prevent overflow
    size_t B;
    size_t T;
    // input handling and its state
    FILE* tokens_file;
    long file_size;
    long current_position;
    // output memory
    int* batch;
    int* inputs;
    int* targets;
    // convenience variables
    size_t num_batches;
} DataLoader;

void dataloader_init(DataLoader *loader, const MultiGpuConfig* multi_gpu_config, const char* filename, size_t B, size_t T) {
    loader->process_rank = multi_gpu_config->process_rank;
    loader->num_processes = multi_gpu_config->num_processes;
    loader->B = B;
    loader->T = T;

    // open the input file for reading
    loader->tokens_file = fopenCheck(filename, "rb");

    // determine the file size
    fseekCheck(loader->tokens_file, 0, SEEK_END);
    loader->file_size = ftell(loader->tokens_file);
    fseekCheck(loader->tokens_file, 0, SEEK_SET);
    if (loader->file_size < (B * T + 1) * sizeof(int)) {
        printf("Error: file size is too small for the batch size and sequence length\n");
        exit(EXIT_FAILURE);
    }
    loader->current_position = loader->process_rank * B * T * sizeof(int); // start at the beginning

    // allocate space for B*T + 1 integers to store the inputs and targets
    // Using CUDA CPU pinned memory for faster PCI Express transfers to GPU
    // See: https://developer.nvidia.com/blog/how-optimize-data-transfers-cuda-cc/
    hipHostMalloc((void**)&loader->batch, (B * T + 1) * sizeof(int));
    loader->inputs = loader->batch;
    loader->targets = loader->batch + 1; // targets are shifted by one
    // note: we definitely want to advance by B * T; That is the "stride" by which we move
    // the window of tokens. We only load B * T + 1 tokens because our targets are offset by 1
    loader->num_batches = loader->file_size / (loader->num_processes * B * T * sizeof(int));
}

void dataloader_reset(DataLoader *loader) {
    loader->current_position = 0;
}

void dataloader_next_batch(DataLoader *loader) {
    size_t B = loader->B;
    size_t T = loader->T;
    // if we are at the end of the file, loop back to the beginning
    if (loader->current_position + (loader->num_processes * B * T + 1) * sizeof(int) > loader->file_size) {
        loader->current_position = loader->process_rank * B * T * sizeof(int);
    }
    // read the B*T+1 integers from the file into batch
    fseekCheck(loader->tokens_file, loader->current_position, SEEK_SET);
    freadCheck(loader->batch, sizeof(int), B*T+1, loader->tokens_file);
    // advance the current position by B*T*num_processes integers
    // note: the "stride" of tokens by which we move each time is definitely B * T
    loader->current_position += loader->num_processes * B * T * sizeof(int);
}

void dataloader_free(DataLoader *loader) {
    fcloseCheck(loader->tokens_file);
    hipHostFree(loader->batch);
}

// ----------------------------------------------------------------------------
// sampler: takes probabilities and samples integers from them

int sample_softmax(const float* logits, int n, float coin) {
    // sample index from logits (converted to probabilities using softmax)
    // coin is a random number in [0, 1), usually from random_f32()
    double norm = 0;
    for (int i = 0; i < n; i++) {
        norm += expf(logits[i]);
    }
    // instead of dividing all exp(logits), we can just multiply coin.
    coin *= norm;
    float cdf = 0.0f;
    for (int i = 0; i < n; i++) {
        cdf += expf(logits[i]);
        if (coin < cdf) {
            return i;
        }
    }
    return n - 1; // in case of rounding errors
}

// ----------------------------------------------------------------------------
// Logger lite, will probably grow/change some over time

typedef struct {
    FILE *logfile;
    int flush_every; // every how many steps to flush the log
} Logger;

void logger_init(Logger *logger, const char *filename) {
    logger->flush_every = 20;
    logger->logfile = NULL;
    if (filename != NULL) { logger->logfile = fopenCheck(filename, "w"); }
}

void logger_log_val(Logger *logger, int step, float val_loss) {
    if (logger->logfile != NULL) {
        fprintf(logger->logfile, "s:%d tel:%.4f\n", step, val_loss);
    }
}

void logger_log_train(Logger *logger, int step, float train_loss) {
    if (logger->logfile != NULL) {
        fprintf(logger->logfile, "s:%d trl:%.4f\n", step, train_loss);
        if (step % 10 == 0) { fflush(logger->logfile); }
    }
}

void logger_free(Logger *logger) {
    if (logger->logfile != NULL) { fclose(logger->logfile); }
}

// ----------------------------------------------------------------------------
// CLI, poor man's argparse

void error_usage() {
    // default run = debugging run with TinyShakespeare
    // bigger run = train on TinyStories! e.g. val/sample less often, but sample more tokens, write to logfile
    fprintf(stderr, "Usage:   ./train_gpt2cu [options]\n");
    fprintf(stderr, "Example: ./train_gpt2cu -i data/TinyStories -v 100 -s 100 -g 144 -o stories.log\n");
    fprintf(stderr, "Options:\n");
    fprintf(stderr, "  -i <string> input dataset prefix (default = data/tiny_shakespeare)\n");
    fprintf(stderr, "  -o <string> output log file (default = NULL)\n");
    fprintf(stderr, "  -b <int>    batch size B (default = 4)\n");
    fprintf(stderr, "  -t <int>    sequence length T (default = 1024)\n");
    fprintf(stderr, "  -l <float>  learning rate (default = 3e-4f)\n");
    fprintf(stderr, "  -x <int>    max_steps of optimization to run (-1 (default) = disable, run 1 epoch)\n");
    fprintf(stderr, "  -v <int>    val_loss_every, how often we evaluate val loss (default = 20)\n");
    fprintf(stderr, "  -m <int>    val_max_batches, up to how many val batches to estimate val loss? (default = 20)\n");
    fprintf(stderr, "  -s <int>    sample_every, how often we inference the model (default = 20)\n");
    fprintf(stderr, "  -g <int>    genT, how many steps of inference we do (default = 64)\n");
    fprintf(stderr, "  -a <int>    overfit a single batch? 0/1. useful for debugging\n");
    fprintf(stderr, "  -f <int>    enable_tf32 override (default: 1, set to 0 to disable tf32)\n");
    fprintf(stderr, "  -w <int>    keep f32 copy of weights for the optimizer? (default: 1)\n");
    exit(EXIT_FAILURE);
}

// ----------------------------------------------------------------------------
// main training loop
int main(int argc, char *argv[]) {
    multi_gpu_config = multi_gpu_config_init(&argc, &argv);

    // read in the (optional) command line arguments
    const char* input_dataset_prefix = "data/tiny_shakespeare"; // or e.g. data/TinyStories
    const char* output_log_file = NULL;
    int B = 4; // batch size
    int T = 1024; // sequence length max
    float learning_rate = 3e-4f;
    int val_loss_every = 20; // every how many steps do we eval validation loss?
    int val_max_batches = 20; // how many batches max do we eval for validation loss?
    int sample_every = 20; // every how many steps to do inference?
    int genT = 64; // number of steps of inference we will do
    int overfit_single_batch = 0; // useful for debugging, 1 = only load a single data batch once
    int max_steps = -1;
    int override_enable_tf32 = 1;
    int use_master_weights = 1;
    for (int i = 1; i < argc; i+=2) {
        if (i + 1 >= argc) { error_usage(); } // must have arg after flag
        if (argv[i][0] != '-') { error_usage(); } // must start with dash
        if (strlen(argv[i]) != 2) { error_usage(); } // must be -x (one dash, one letter)
        // read in the args
        if (argv[i][1] == 'i') { input_dataset_prefix = argv[i+1]; }
        else if (argv[i][1] == 'o') { output_log_file = argv[i+1]; }
        else if (argv[i][1] == 'b') { B = atoi(argv[i+1]); } // Per-GPU batch size
        else if (argv[i][1] == 't') { T = atoi(argv[i+1]); }
        else if (argv[i][1] == 'l') { learning_rate = atof(argv[i+1]); }
        else if (argv[i][1] == 'x') { max_steps = atoi(argv[i+1]); }
        else if (argv[i][1] == 'v') { val_loss_every = atoi(argv[i+1]); }
        else if (argv[i][1] == 'm') { val_max_batches = atoi(argv[i+1]); }
        else if (argv[i][1] == 's') { sample_every = atoi(argv[i+1]); }
        else if (argv[i][1] == 'g') { genT = atoi(argv[i+1]); }
        else if (argv[i][1] == 'a') { overfit_single_batch = atoi(argv[i+1]); }
        else if (argv[i][1] == 'f') { override_enable_tf32 = atoi(argv[i+1]); }
        else if (argv[i][1] == 'w') { use_master_weights = atoi(argv[i+1]); }
        else { error_usage(); }
    }
    printf0("+-----------------------+----------------------------------------------------+\n");
    printf0("| Parameter             | Value                                              |\n");
    printf0("+-----------------------+----------------------------------------------------+\n");
    printf0("| input dataset prefix  | %-50s |\n", input_dataset_prefix);
    printf0("| output log file       | %-50s |\n", output_log_file == NULL ? "NULL" : output_log_file);
    printf0("| batch size B          | %-50d |\n", B);
    printf0("| sequence length T     | %-50d |\n", T);
    printf0("| learning rate         | %-50e |\n", learning_rate);
    printf0("| max_steps             | %-50d |\n", max_steps);
    printf0("| val_loss_every        | %-50d |\n", val_loss_every);
    printf0("| val_max_batches       | %-50d |\n", val_max_batches);
    printf0("| sample_every          | %-50d |\n", sample_every);
    printf0("| genT                  | %-50d |\n", genT);
    printf0("| overfit_single_batch  | %-50d |\n", overfit_single_batch);
    printf0("| use_master_weights    | %-50s |\n", use_master_weights ? "enabled" : "disabled");
    printf0("+-----------------------+----------------------------------------------------+\n");

    // set up the device
    cudaCheck(hipSetDevice(multi_gpu_config.local_device_idx));
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, multi_gpu_config.local_device_idx);
    cuda_num_SMs = deviceProp.multiProcessorCount;
    cuda_arch_major = deviceProp.major;
    cuda_arch_minor = deviceProp.minor;

    // set up cuBLAS and cuBLASLt
    cublasCheck(hipblasCreate(&cublas_handle));
    cublasCheck(hipblasLtCreate(&cublaslt_handle));
    cudaCheck(hipMalloc(&cublaslt_workspace, cublaslt_workspace_size));
    // setup compute precision settings for cublas
    // TF32 precision is equivalent to torch.set_float32_matmul_precision('high')
    int enable_tf32 = cuda_arch_major >= 8 ? 1 : 0;
    if (override_enable_tf32 == 0) { enable_tf32 = 0; } // force to zero via arg
    cublas_compute_type = enable_tf32 ? HIPBLAS_COMPUTE_32F_FAST_TF32 : HIPBLAS_COMPUTE_32F;
    hipblasMath_t cublas_math_mode = enable_tf32 ? HIPBLAS_TF32_TENSOR_OP_MATH : HIPBLAS_DEFAULT_MATH;
    cublasCheck(hipblasSetMathMode(cublas_handle, cublas_math_mode));
    if(cublas_compute_type); // unused in BF16 mode, avoid warning

    // set up cuDNN
    #ifdef ENABLE_CUDNN
    checkCudnnErr(hipdnnCreate(&cudnn_handle));
    #endif

    printf0("| device                | %-50s |\n", deviceProp.name);
    printf0("| TF32                  | %-50s |\n", enable_tf32 ? "enabled" : "disabled");
    printf0("| precision             | %-50s |\n", precision_mode_str);
    printf0("+-----------------------+----------------------------------------------------+\n");

    // build the GPT-2 model from a checkpoint
    GPT2 model;
    gpt2_build_from_checkpoint(&model, load_filename);
    model.use_master_weights = use_master_weights;
    printf0("| load_filename         | %-50s |\n", load_filename);
    printf0("| max_sequence_length T | %-50d |\n", model.config.max_seq_len);
    printf0("| vocab_size V          | %-50d |\n", model.config.vocab_size);
    printf0("| padded_vocab_size Vp  | %-50d |\n", model.config.padded_vocab_size);
    printf0("| num_layers L          | %-50d |\n", model.config.num_layers);
    printf0("| num_heads NH          | %-50d |\n", model.config.num_heads);
    printf0("| channels C            | %-50d |\n", model.config.channels);
    printf0("| num_parameters        | %-50zu |\n", model.num_parameters);
    printf0("+-----------------------+----------------------------------------------------+\n");

    // build DataLoaders for both train and val
    char train_tokens_filename[128];
    char val_tokens_filename[128];
    assert(strlen(input_dataset_prefix) < 100); // being bit lazy here, make sure we don't overflow
    // if we're only overfitting a single batch for debugging, let's overfit the first batch
    // from val instead of train split, because val is smaller and a bit faster
    const char* train_split = (overfit_single_batch == 1) ? "val" : "train";
    sprintf(train_tokens_filename, "%s_%s.bin", input_dataset_prefix, train_split);
    sprintf(val_tokens_filename, "%s_val.bin", input_dataset_prefix);
    DataLoader train_loader;
    dataloader_init(&train_loader, &multi_gpu_config, train_tokens_filename, B, T);
    DataLoader val_loader;
    dataloader_init(&val_loader, &multi_gpu_config, val_tokens_filename, B, T);
    int train_num_batches = (max_steps == -1) ? train_loader.num_batches : max_steps; // default = 1 epoch
    int val_num_batches = train_loader.num_batches < val_max_batches ? train_loader.num_batches : val_max_batches;
    printf0("| train_num_batches     | %-50d |\n", train_num_batches);
    printf0("| val_num_batches       | %-50d |\n", val_num_batches);
    printf0("+-----------------------+----------------------------------------------------+\n");

    // pretty print in a table the multi-gpu configuration as well
    printf0("| num_processes         | %-50d |\n", multi_gpu_config.num_processes);
    printf0("+-----------------------+----------------------------------------------------+\n");

    // more prints related to allocations from gpt2_build_from_checkpoint down here to not mess up our table above
    printf0("num_parameters: %zu ==> bytes: %zu\n", model.num_parameters, model.num_parameters_bytes);
    printf0("allocated %d MiB for model parameters\n", (int)round(model.num_parameters_bytes / (1024 * 1024)));

    // set up the Logger
    Logger logger;
    logger_init(&logger, output_log_file);

    // build the Tokenizer
    Tokenizer tokenizer;
    tokenizer_init(&tokenizer, "gpt2_tokenizer.bin");

    // some memory for generating samples from the model
    unsigned long long rng_state = 1337;
    int* gen_tokens = (int*)mallocCheck(B * T * sizeof(int));
    floatX* cpu_logits_raw = (floatX*)mallocCheck(model.config.vocab_size * sizeof(floatX));
    float*  cpu_logits = (float*)mallocCheck(model.config.vocab_size * sizeof(float));

    // train
    struct timespec start, end;
    double total_sum_iteration_time_s = 0.0;
    for (int step = 0; step <= train_num_batches; step++) {
        int last_step = step == train_num_batches;

        // once in a while estimate the validation loss
        if (step % val_loss_every == 0 || last_step) {
            float val_loss = 0.0f;
            dataloader_reset(&val_loader);
            for (int i = 0; i < val_num_batches; i++) {
                dataloader_next_batch(&val_loader);
                gpt2_forward(&model, val_loader.inputs, val_loader.targets, B, T);
                val_loss += model.mean_loss;
            }
            val_loss /= val_num_batches;
            val_loss = multi_gpu_cpu_float_mean(val_loss, &multi_gpu_config);
            printf0("val loss %f\n", val_loss);
            logger_log_val(&logger, step, val_loss);
        }

        // once in a while do model inference to print generated text
        if (multi_gpu_config.process_rank == 0 && (step > 0 && (step % sample_every) == 0 || last_step)) {
            // fill up gen_tokens with the <|endoftext|> token, which kicks off the generation
            int eot_token = tokenizer.eot_token;
            for(int i = 0; i < B * T; ++i) {
                gen_tokens[i] = eot_token;
            }
            // now sample from the model autoregressively
            printf("generating:\n---\n");
            for (int t = 1; t < genT; t++) {
                // note that inference is very wasteful here because for each token
                // we re-calculate the forward pass for all of (B,T) positions from scratch
                // but the inference here is just for sanity checking anyway
                // and we can maybe optimize a bit more later, with careful tests
                gpt2_forward(&model, gen_tokens, NULL, B, T);
                // furthermore, below we're only using b=0 (i.e. the first row) of all B rows
                // we're in principle running B "inference streams" in parallel here
                // only using position 0 because it's a bit faster (copy less probs from GPU -> CPU)
                // get the V-dimensional vector probs[0, t-1, :]
                floatX* logits = model.acts.output + (t - 1) * model.config.padded_vocab_size;
                // move probs back to CPU and sample (note we only move the first vocab_size logits, ignoring the padding)
                cudaCheck(hipMemcpy(cpu_logits_raw, logits, model.config.vocab_size * sizeof(floatX), hipMemcpyDeviceToHost));
                // convert to FP32 into cpu_logits (this does nothing useful if floatX == float)
                for (int i = 0; i < model.config.vocab_size; i++) {
                    cpu_logits[i] = (float)cpu_logits_raw[i];
                }

                float coin = random_f32(&rng_state);
                int next_token = sample_softmax(cpu_logits, model.config.vocab_size, coin);
                gen_tokens[t] = next_token;
                // print the generated token, either using the Tokenizer or a fallback
                if (tokenizer.init_ok) {
                    const char* token_str = tokenizer_decode(&tokenizer, next_token);
                    safe_printf(token_str);
                } else {
                    // fall back to printing the token id
                    printf("%d ", next_token);
                }
                fflush(stdout);
            }
            printf("\n---\n");
        }

        // bit confusing: we want to make sure to eval and sample on 0th iteration
        // but also after the very last iteration. so we loop for step <= train_num_batches
        // instead of just < train_num_batches (one extra due to <=), only to do
        // the validation/sampling one last time, and then we break right here as we're done.
        if (last_step) { break; }

        // do a training step
        clock_gettime(CLOCK_MONOTONIC, &start);
        if (overfit_single_batch == 0 || (step == 0 && overfit_single_batch == 1)) {
            // if we're overfitting a single batch, we'll only call this at step = 0
            dataloader_next_batch(&train_loader);
        }
        gpt2_forward(&model, train_loader.inputs, train_loader.targets, B, T);
        gpt2_zero_grad(&model);
        gpt2_backward(&model);
        if (multi_gpu_config.num_processes > 1) {
            gpt2_multi_gpu_accumulate(&model, &multi_gpu_config);
        }
        gpt2_update(&model, learning_rate, 0.9f, 0.999f, 1e-8f, 0.0f, step+1);
        cudaCheck(hipDeviceSynchronize()); // finish all CUDA work to get correct precise timings
        clock_gettime(CLOCK_MONOTONIC, &end);
        double time_elapsed_s = (end.tv_sec - start.tv_sec) + (end.tv_nsec - start.tv_nsec) / 1e9;

        if (step > 0) { // consider the first batch to be a warmup (e.g. cuBLAS/cuDNN initialisation)
            total_sum_iteration_time_s += time_elapsed_s;
        }
        int tokens_per_second = multi_gpu_config.num_processes * (B * T) / time_elapsed_s;
        float accumulated_loss = multi_gpu_config.num_processes == 1 ? model.mean_loss : model.accumulated_mean_loss;
        printf0("step %4d/%d: train loss %f (acc %f) (%f ms, %d tok/s)\n", step + 1, train_num_batches, model.mean_loss, accumulated_loss, time_elapsed_s * 1000, tokens_per_second);
        logger_log_train(&logger, step, model.mean_loss);
    }
    // add a total average, for optimizations that are only mild improvements (excluding 1st batch as warmup)
    printf0("total average iteration time: %f ms\n", total_sum_iteration_time_s / (train_num_batches-1) * 1000);

    // free and destroy everything
    dataloader_free(&train_loader);
    dataloader_free(&val_loader);
    tokenizer_free(&tokenizer);
    gpt2_free(&model);
    free(cpu_logits_raw);
    free(cpu_logits);
    free(gen_tokens);
    #ifdef ENABLE_CUDNN
    if (cudnn_workspace != NULL) { cudaCheck(hipFree(cudnn_workspace)); }
    checkCudnnErr(hipdnnDestroy(cudnn_handle));
    #endif
    cudaCheck(hipFree(cublaslt_workspace));
    cublasCheck(hipblasDestroy(cublas_handle));
    cublasCheck(hipblasLtDestroy(cublaslt_handle));
    logger_free(&logger);
    multi_gpu_config_free(&multi_gpu_config);

    return 0;
}
#endif
